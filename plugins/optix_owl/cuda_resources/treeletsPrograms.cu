#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>

#include <owl/common/math/box.h>
#include <owl/common/math/vec.h>
#include <owl/owl_device.h>

#include "intersect.h"
#include "perraydata.h"
#include "treelets.h"

namespace megamol {
namespace optix_owl {
namespace device {
using namespace owl::common;

struct StackEntry {
    float t0, t1;
    unsigned int nodeID;
};

OPTIX_INTERSECT_PROGRAM(treelet_brute_intersect)() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<TreeletsGeomData>();
    const auto treelet = self.treeletBuffer[treeletID];

    owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    const int begin = treelet.begin;
    float tmp_hit_t = ray.tmax;
    int tmp_hit_primID = -1;
    for (int particleID = begin; particleID < treelet.end; ++particleID) {
        const Particle particle = self.particleBuffer[particleID];
        if (intersectSphere(particle, self.particleRadius, ray, tmp_hit_t))
            tmp_hit_primID = particleID;
    }
    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax)
        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
}

OPTIX_INTERSECT_PROGRAM(treelets_intersect)() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<TreeletsGeomData>();
    const auto treelet = self.treeletBuffer[treeletID];

    owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        if (!clipToBounds(ray, treelet.bounds, t0, t1))
            return;


        int nodeID = 0;
        float tmp_hit_t = t1;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };
        StackEntry stackBase[STACK_DEPTH];
        StackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;
                const Particle particle = self.particleBuffer[particleID];
                int const dim = particle.get_dim();

                const float t_slab_lo = (particle.pos[dim] - self.particleRadius - org[dim]) * rdir[dim];
                const float t_slab_hi = (particle.pos[dim] + self.particleRadius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(particle, self.particleRadius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;
                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);
                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}

OPTIX_CLOSEST_HIT_PROGRAM(treelets_ch)() {
    PerRayData& prd = owl::getPRD<PerRayData>();
    const auto& self = owl::getProgramData<TreeletsGeomData>();
    prd.particleID = optixGetAttribute_0();
    prd.t = optixGetRayTmax();
    prd.pos = self.particleBuffer[optixGetAttribute_0()].pos;
}

OPTIX_BOUNDS_PROGRAM(treelets_bounds)(const void* geomData, box3f& primBounds, const int primID) {
    primBounds = ((const TreeletsGeomData*) geomData)->treeletBuffer[primID].bounds;
}
} // namespace device
} // namespace optix_owl
} // namespace megamol
