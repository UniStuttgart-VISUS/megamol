#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>

#include <owl/common/math/box.h>
#include <owl/common/math/vec.h>
#include <owl/owl_device.h>

#include "intersect.h"
#include "perraydata.h"
#include "floatcomp.h"

namespace megamol {
namespace optix_owl {
namespace device {
using namespace owl::common;

struct FloatCompStackEntry {
    float t0, t1;
    unsigned int nodeID;
};

struct FloatCompStackEntryDep {
    float t0, t1;
    unsigned int nodeID;
    vec3f refPos;
};

inline __device__ vec3f getParticle(
    QTParticle_e5m15 const& data, int& dim, char const* exp_x_lu, char const* exp_y_lu, char const* exp_z_lu) {
    vec3f pos;

    unsigned int x = 0;
    char exp_x = exp_x_lu[data.exp_x];
    x += ((int) exp_x + 127u) << 23;
    x += (((unsigned int) data.m_x) << QTParticle_e5m15::offset);

    unsigned int y = 0;
    char exp_y = exp_y_lu[data.exp_y];
    y += ((int) exp_y + 127u) << 23;
    y += (((unsigned int) data.m_y) << QTParticle_e5m15::offset);

    unsigned int z = 0;
    char exp_z = exp_z_lu[data.exp_z];
    z += ((int) exp_z + 127u) << 23;
    z += (((unsigned int) data.m_z) << QTParticle_e5m15::offset);

    pos.x = __uint_as_float(x);
    pos.y = __uint_as_float(y);
    pos.z = __uint_as_float(z);

    if (data.dim_x)
        dim = 0;
    if (data.dim_y)
        dim = 1;
    if (data.dim_z)
        dim = 2;

    return pos;
}
inline __device__ int getDim(QTParticle_e5m15 const& data) {
    if (data.dim_x)
        return 0;
    if (data.dim_y)
        return 1;
    if (data.dim_z)
        return 2;
}

inline __device__ vec3f getParticle(QTParticle_e5m15d const& data, bool left_child, int sep_dim, int& dim,
    char const* exp_x_lu, char const* exp_y_lu, char const* exp_z_lu) {

    dim = (data.dim_b << 1) + data.dim_a;

    vec3f pos;

    unsigned int sign_x = sep_dim == 0 ? left_child : data.sign_a;
    unsigned int sign_y = sep_dim == 1 ? left_child : (sep_dim == 0 ? data.sign_a : data.sign_b);
    unsigned int sign_z = sep_dim == 2 ? left_child : data.sign_b;

    unsigned int x = 0;
    x += (sign_x) << 31;
    char exp_x = exp_x_lu[data.exp_x];
    x += ((int) exp_x + 127u) << 23;
    x += (((unsigned int) data.m_x) << QTParticle_e5m15d::offset);

    unsigned int y = 0;
    y += (sign_y) << 31;
    char exp_y = exp_y_lu[data.exp_y];
    y += ((int) exp_y + 127u) << 23;
    y += (((unsigned int) data.m_y) << QTParticle_e5m15d::offset);

    unsigned int z = 0;
    z += (sign_z) << 31;
    char exp_z = exp_z_lu[data.exp_z];
    z += ((int) exp_z + 127u) << 23;
    z += (((unsigned int) data.m_z) << QTParticle_e5m15d::offset);

    pos.x = __uint_as_float(x);
    pos.y = __uint_as_float(y);
    pos.z = __uint_as_float(z);

    return pos;
}
inline __device__ int getDim(QTParticle_e5m15d const& data) {
    return (data.dim_b << 1) + data.dim_a;
}

template<typename BUF_TYPE, int BEXP = BUF_TYPE::exp>
void __device__ traverse_dep(BUF_TYPE const* buffer) {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<FloatCompGeomData>();
    const auto& treelet = self.treeletBuffer[treeletID];
    PerRayData& prd = owl::getPRD<PerRayData>();
    //auto const localTables = self.localTables;

    int const num_idx = powf(2, BEXP);

    char const* exp_x_lu = self.expXBuffer;
    char const* exp_y_lu = self.expYBuffer;
    char const* exp_z_lu = self.expZBuffer;

    if (self.use_localtables > 0) {
        exp_x_lu += treeletID * num_idx;
        exp_y_lu += treeletID * num_idx;
        exp_z_lu += treeletID * num_idx;
    }

    //#ifdef LOCAL_TABLES
    //    if (treelet.has_local_tables) {
    //        exp_x_lu = treelet.exp_x;
    //        exp_y_lu = treelet.exp_y;
    //        exp_z_lu = treelet.exp_z;
    //    }
    //#endif


    //const auto& test_ref = self.qtpBuffer[0];

    auto const ray = owl::Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        if (!clipToBounds(ray, treelet.bounds, t0, t1))
            return;


        int nodeID = 0;
        float tmp_hit_t = t1;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };

        FloatCompStackEntryDep stackBase[STACK_DEPTH];
        FloatCompStackEntryDep* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };


        vec3f pos;
        int dim;
        vec3f refPos = treelet.basePos;


        vec3f tmp_hit_pos;

        //const float radius = self.radius;

        //unsigned int baseOffset = 0;
        /*if (localTables) {
            baseOffset = 1;
        }*/

        //auto const* buffer = (QTParticle_e4m16d const*)self.qtpBuffer;
        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;

                //auto test = self.qtpBuffer[particleID];

                //getParticle(treelet, self.qtpBuffer[particleID], dim, pos, exp_x_lu, exp_y_lu, exp_z_lu);

                /*if (nodeID == 0) {
                    pos = refPos;
                    dim = getDim(buffer[particleID]);
                } else*/
                {
                    auto const parentID = parent(nodeID) + begin;

                    //unsigned int offset = treeletID * num_idx * baseOffset;
                    pos = getParticle(buffer[particleID], nodeID % 2 == 1, getDim(buffer[parentID]), dim,
                              exp_x_lu /*+ offset*/, exp_y_lu /*+ offset*/, exp_z_lu /*+ offset*/) +
                          refPos;

                    /*pos = getParticle(buffer[particleID], nodeID % 2 == 1,
                        #ifdef E4M16
                        buffer[parentID].dim,
                        #else
                              (buffer[parentID].dim_b << 1) + buffer[parentID].dim_a,
                        #endif
                                          dim, exp_x_lu + offset, exp_y_lu + offset, exp_z_lu + offset) +
                                      refPos;*/
                }

                //getParticle(treelet, self.qparticleBuffer[particleID], dim, pos);
                /*const pkd::Particle particle = self.particleBuffer[particleID];
                int const dim = particle.dim;*/
                //pos = pos * refPos;

                const float t_slab_lo = (pos[dim] - self.particleRadius - org[dim]) * rdir[dim];
                const float t_slab_hi = (pos[dim] + self.particleRadius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(pos, self.particleRadius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;

                        tmp_hit_pos = pos;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + dir_sign[dim];
                const int farSide_nodeID = 2 * nodeID + 2 - dir_sign[dim];

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    //setEntry(stackPtr, farSide_nodeID, farSide_t0, farSide_t1);

                    stackPtr->refPos = pos;

                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;

                    refPos = pos;


                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;

                refPos = pos;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        prd.pos = tmp_hit_pos;
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                //getEntry(stackPtr, nodeID, t0, t1);
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);

                refPos = stackPtr->refPos;

                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}

template<typename BUF_TYPE, int BEXP = BUF_TYPE::exp>
void __device__ traverse(BUF_TYPE const* buffer) {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<FloatCompGeomData>();
    const auto& treelet = self.treeletBuffer[treeletID];
    PerRayData& prd = owl::getPRD<PerRayData>();
    //auto const localTables = self.localTables;

    int const num_idx = powf(2, BEXP);

    char const* exp_x_lu = self.expXBuffer;
    char const* exp_y_lu = self.expYBuffer;
    char const* exp_z_lu = self.expZBuffer;

    if (self.use_localtables > 0) {
        exp_x_lu += treeletID * num_idx;
        exp_y_lu += treeletID * num_idx;
        exp_z_lu += treeletID * num_idx;
    }

    auto const ray = owl::Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        if (!clipToBounds(ray, treelet.bounds, t0, t1))
            return;


        int nodeID = 0;
        float tmp_hit_t = t1;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };

        FloatCompStackEntry stackBase[STACK_DEPTH];
        FloatCompStackEntry* stackPtr = stackBase;

        const int dir_sign[3] = {ray.direction.x < 0.f, ray.direction.y < 0.f, ray.direction.z < 0.f};
        const float org[3] = {ray.origin.x, ray.origin.y, ray.origin.z};
        const float rdir[3] = {
            (fabsf(ray.direction.x) <= 1e-8f) ? 1e8f : 1.f / ray.direction.x,
            (fabsf(ray.direction.y) <= 1e-8f) ? 1e8f : 1.f / ray.direction.y,
            (fabsf(ray.direction.z) <= 1e-8f) ? 1e8f : 1.f / ray.direction.z,
        };


        vec3f pos;
        int dim;
        vec3f refPos = treelet.basePos;

        vec3f tmp_hit_pos;


        //const float radius = self.radius;

        //unsigned int baseOffset = 0;
        /*if (localTables) {
            baseOffset = 1;
        }*/

        //auto const* buffer = (QTParticle_e4m16 const*)self.qtpBuffer;
        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;

                //auto test = self.qtpBuffer[particleID];

                //getParticle(treelet, self.qtpBuffer[particleID], dim, pos, exp_x_lu, exp_y_lu, exp_z_lu);

                //unsigned int offset = treeletID * num_idx * baseOffset;
                pos = getParticle(buffer[particleID], dim, exp_x_lu /*+ offset*/, exp_y_lu /*+ offset*/,
                          exp_z_lu /*+ offset*/) +
                      refPos;


                //getParticle(treelet, self.qparticleBuffer[particleID], dim, pos);
                /*const pkd::Particle particle = self.particleBuffer[particleID];
                int const dim = particle.dim;*/
                //pos = pos * refPos;

                const float t_slab_lo = (pos[dim] - self.particleRadius - org[dim]) * rdir[dim];
                const float t_slab_hi = (pos[dim] + self.particleRadius - org[dim]) * rdir[dim];

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                if (sphere_t0 < sphere_t1) {
                    if (intersectSphere(pos, self.particleRadius, ray, tmp_hit_t)) {
                        tmp_hit_primID = particleID;
                        tmp_hit_pos = pos;
                    }
                }

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + (ray.direction[dim] < 0.f);
                const int farSide_nodeID = 2 * nodeID + 2 - (ray.direction[dim] < 0.f);

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;
                    //setEntry(stackPtr, farSide_nodeID, farSide_t0, farSide_t1);

                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;


                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        prd.pos = tmp_hit_pos;
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                //getEntry(stackPtr, nodeID, t0, t1);
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = fminf(t1, tmp_hit_t);

                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}

OPTIX_INTERSECT_PROGRAM(floatcomp_intersect_e5m15d)() {
    const auto& self = owl::getProgramData<FloatCompGeomData>();
    auto const* buffer = (QTParticle_e5m15d const*) self.particleBuffer;
    traverse_dep(buffer);
}

OPTIX_INTERSECT_PROGRAM(floatcomp_intersect_e5m15)() {
    const auto& self = owl::getProgramData<FloatCompGeomData>();
    auto const* buffer = (QTParticle_e5m15 const*) self.particleBuffer;
    traverse(buffer);
}

OPTIX_CLOSEST_HIT_PROGRAM(floatcomp_ch)() {
    PerRayData& prd = owl::getPRD<PerRayData>();
    const auto& self = owl::getProgramData<FloatCompGeomData>();
    prd.particleID = optixGetAttribute_0();
    prd.t = optixGetRayTmax();
}

OPTIX_BOUNDS_PROGRAM(floatcomp_bounds)(const void* geomData, box3f& primBounds, const int primID) {
    primBounds = ((const FloatCompGeomData*) geomData)->treeletBuffer[primID].bounds;
}
} // namespace device
} // namespace optix_owl
} // namespace megamol
