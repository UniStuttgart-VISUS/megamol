// code originally from https://github.com/UniStuttgart-VISUS/rtxpkd_ldav2020
// modified for MegaMol

// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// ======================================================================== //
// Modified 2019-2025 VISUS - University of Stuttgart                       //
// ======================================================================== //

#include <hip/hip_runtime.h>
#include <optix_device.h>

#include <owl/common/math/box.h>
#include <owl/common/math/vec.h>
#include <owl/owl_device.h>

#include "intersect.h"
#include "perraydata.h"
#include "bvh.h"

namespace megamol {
namespace optix_owl {
namespace device {
using namespace owl::common;

OPTIX_INTERSECT_PROGRAM(bvh_intersect)() {
    const int primID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<BVHGeomData>();

    owl::Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    float tmp_hit_t = ray.tmax;

    const Particle& particle = self.particleBuffer[primID];
    if (intersectSphere(particle, self.particleRadius, ray, tmp_hit_t))
        optixReportIntersection(tmp_hit_t, 0);
}

OPTIX_CLOSEST_HIT_PROGRAM(bvh_ch)() {
    const int primID = optixGetPrimitiveIndex();
    PerRayData& prd = owl::getPRD<PerRayData>();
    const auto& self = owl::getProgramData<BVHGeomData>();
    prd.particleID = primID;
    prd.t = optixGetRayTmax();
    prd.pos = self.particleBuffer[primID].pos;
}

OPTIX_BOUNDS_PROGRAM(bvh_bounds)(const void* geomData, box3f& primBounds, const int primID) {
    auto& self = *(const BVHGeomData*) geomData;
    primBounds.lower = self.particleBuffer[primID].pos - self.particleRadius;
    primBounds.upper = self.particleBuffer[primID].pos + self.particleRadius;
}
} // namespace device
} // namespace optix_owl
} // namespace megamol
