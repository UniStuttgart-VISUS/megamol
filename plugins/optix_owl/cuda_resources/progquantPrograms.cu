#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <optix_device.h>

#include <owl/common/math/box.h>
#include <owl/common/math/vec.h>
#include <owl/owl_device.h>

#include "intersect.h"
#include "perraydata.h"
#include "progquant.h"

namespace megamol {
namespace optix_owl {
namespace device {
using namespace owl::common;

struct ProgQuantStackEntry {
    float t0, t1;
    unsigned int nodeID;
    box3f refBox;
};

OPTIX_INTERSECT_PROGRAM(progquant_intersect)() {
    const int treeletID = optixGetPrimitiveIndex();
    const auto& self = owl::getProgramData<ProgQuantGeomData>();
    const auto treelet = self.treeletBuffer[treeletID];
    PerRayData& prd = owl::getPRD<PerRayData>();

    auto const ray = owl::Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

    const int begin = treelet.begin;
    const int size = treelet.end - begin;
    {
        float t0, t1;
        if (!clipToBounds(ray, treelet.bounds, t0, t1))
            return;


        int nodeID = 0;
        float tmp_hit_t = ray.tmax;
        int tmp_hit_primID = -1;

        enum { STACK_DEPTH = 12 };

        ProgQuantStackEntry stackBase[STACK_DEPTH];
        ProgQuantStackEntry* stackPtr = stackBase;

        vec3f pos;
        int dim;

        box3f refBox = treelet.bounds;

        vec3f tmp_hit_pos;

        float compensation = 0.f;

        while (1) {
            // while we have anything to traverse ...

            while (1) {
                // while we can go down

                const int particleID = nodeID + begin;

                {
                    auto const& bpart = self.particleBuffer[particleID];
                    dim = bpart.dim;
                    pos = bpart.from(refBox.span(), refBox.lower);
                }

                compensation = t_compensate(refBox.span()[dim]);

                const float t_slab_lo = (pos[dim] - self.particleRadius - ray.origin[dim]) / ray.direction[dim] - compensation;
                const float t_slab_hi = (pos[dim] + self.particleRadius - ray.origin[dim]) / ray.direction[dim] + compensation;

                const float t_slab_nr = fminf(t_slab_lo, t_slab_hi);
                const float t_slab_fr = fmaxf(t_slab_lo, t_slab_hi);

                // -------------------------------------------------------
                // compute potential sphere interval, and intersect if necessary
                // -------------------------------------------------------
                /*const float sphere_t0 = fmaxf(t0, t_slab_nr);
                const float sphere_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t);*/

                //if (sphere_t0 < sphere_t1) {
                if (intersectSphere(pos, self.particleRadius, ray, tmp_hit_t)) {
                    tmp_hit_primID = particleID;

                    tmp_hit_pos = pos;
                }
                //}

                // -------------------------------------------------------
                // compute near and far side intervals
                // -------------------------------------------------------
                const float nearSide_t0 = t0;
                const float nearSide_t1 = fminf(fminf(t_slab_fr, t1), tmp_hit_t - compensation);

                const float farSide_t0 = fmaxf(t0, t_slab_nr);
                const float farSide_t1 = fminf(t1, tmp_hit_t + compensation);

                // -------------------------------------------------------
                // logic
                // -------------------------------------------------------
                const int nearSide_nodeID = 2 * nodeID + 1 + (ray.direction[dim] < 0.f);
                const int farSide_nodeID = 2 * nodeID + 2 - (ray.direction[dim] < 0.f);

                const bool nearSide_valid = nearSide_nodeID < size;
                const bool farSide_valid = farSide_nodeID < size;

                const bool need_nearSide = nearSide_valid && nearSide_t0 < nearSide_t1;
                const bool need_farSide = farSide_valid && farSide_t0 < farSide_t1;

                if (!(need_nearSide || need_farSide))
                    break; // pop ...

                if (need_nearSide && need_farSide) {
                    stackPtr->nodeID = farSide_nodeID;
                    stackPtr->t0 = farSide_t0;
                    stackPtr->t1 = farSide_t1;

                    if (ray.direction[dim] < 0.f) {
                        stackPtr->refBox = leftBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                        refBox = rightBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                    } else {
                        stackPtr->refBox = rightBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                        refBox = leftBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                    }

                    ++stackPtr;

                    nodeID = nearSide_nodeID;
                    t0 = nearSide_t0;
                    t1 = nearSide_t1;

                    continue;
                }

                nodeID = need_nearSide ? nearSide_nodeID : farSide_nodeID;
                t0 = need_nearSide ? nearSide_t0 : farSide_t0;
                t1 = need_nearSide ? nearSide_t1 : farSide_t1;
                if (ray.direction[dim] < 0.f) {
                    refBox = need_nearSide ? rightBounds(refBox, pos[dim], self.particleRadius, dim, compensation)
                                           : leftBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                } else {
                    refBox = need_nearSide ? leftBounds(refBox, pos[dim], self.particleRadius, dim, compensation)
                                           : rightBounds(refBox, pos[dim], self.particleRadius, dim, compensation);
                }
            }
            // -------------------------------------------------------
            // pop
            // -------------------------------------------------------
            while (1) {
                if (stackPtr == stackBase) {
                    // can't pop any more - done.
                    if (tmp_hit_primID >= 0 && tmp_hit_t < ray.tmax) {
                        prd.pos = tmp_hit_pos;
                        optixReportIntersection(tmp_hit_t, 0, tmp_hit_primID);
                    }
                    return;
                }
                --stackPtr;
                //getEntry(stackPtr, nodeID, t0, t1);
                t0 = stackPtr->t0;
                t1 = stackPtr->t1;
                nodeID = stackPtr->nodeID;
                t1 = min(t1, tmp_hit_t);

                refBox = stackPtr->refBox;

                if (t1 <= t0)
                    continue;
                break;
            }
        }
    }
}

OPTIX_CLOSEST_HIT_PROGRAM(progquant_ch)() {
    PerRayData& prd = owl::getPRD<PerRayData>();
    prd.particleID = optixGetAttribute_0();
    prd.t = optixGetRayTmax();
}

OPTIX_BOUNDS_PROGRAM(progquant_bounds)(const void* geomData, box3f& primBounds, const int primID) {
    primBounds = ((const ProgQuantGeomData*) geomData)->treeletBuffer[primID].bounds;
}
} // namespace device
} // namespace optix_owl
} // namespace megamol
