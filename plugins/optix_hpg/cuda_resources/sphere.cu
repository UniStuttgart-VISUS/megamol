#include "hip/hip_runtime.h"
#include "perraydata.h"
#include "sphere.h"

#include "glm/glm.hpp"
#include "optix/utils_device.h"

#include "optix/random.h"

#include "glm/gtx/component_wise.hpp"

// OptiX SDK and rtxpkd

namespace megamol {
namespace optix_hpg {
    namespace device {
        inline __device__ void intersectSphere(const Particle& particle, const float particleRadius, const Ray& ray) {
            // Raytracing Gems Intersection Code (Chapter 7)
            const glm::vec3 pos = glm::vec3(particle.pos);
            const glm::vec3 oc = ray.origin - pos;
            const float sqrRad = particleRadius * particleRadius;

            // const float  a = dot(ray.direction, ray.direction);
            const float b = glm::dot(-oc, ray.direction);
            const glm::vec3 temp = oc + b * ray.direction;
            const float delta = sqrRad - glm::dot(temp, temp);

            if (delta < 0.0f)
                return;

            const float c = glm::dot(oc, oc) - sqrRad;
            const float q = b + copysignf(sqrtf(delta), b);

            {
                const float t = fminf(c / q, q);
                if (t > ray.tmin && t < ray.tmax) {
                    optixReportIntersection(t, 0);
                }
            }
        }

        MM_OPTIX_INTERSECTION_KERNEL(sphere_intersect)() {
            // printf("ISEC\n");

            const int primID = optixGetPrimitiveIndex();

            const auto& self = getProgramData<SphereGeoData>();

            auto const ray =
                Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

            const Particle& particle = self.particleBufferPtr[primID];
            // float tmp_hit_t = ray.tmax;
            /*if (intersectSphere(particle, particle.pos.w, ray, tmp_hit_t)) {
                optixReportIntersection(tmp_hit_t, 0);
            }*/
            intersectSphere(particle, particle.pos.w, ray);
        }

        // OptiX SDK
        // Path tracer example

        //
        // Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
        //
        // Redistribution and use in source and binary forms, with or without
        // modification, are permitted provided that the following conditions
        // are met:
        //  * Redistributions of source code must retain the above copyright
        //    notice, this list of conditions and the following disclaimer.
        //  * Redistributions in binary form must reproduce the above copyright
        //    notice, this list of conditions and the following disclaimer in the
        //    documentation and/or other materials provided with the distribution.
        //  * Neither the name of NVIDIA CORPORATION nor the names of its
        //    contributors may be used to endorse or promote products derived
        //    from this software without specific prior written permission.
        //
        // THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
        // EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
        // IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
        // PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
        // CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
        // EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
        // PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
        // PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
        // OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
        // (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
        // OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
        //

        //
        // Modified 2021 MegaMol Dev Team
        //

        MM_OPTIX_CLOSESTHIT_KERNEL(sphere_closesthit)() {
            const int primID = optixGetPrimitiveIndex();
            PerRayData& prd = getPerRayData<PerRayData>();
            /*prd.primID = primID;
            prd.t = optixGetRayTmax();*/

            const auto& self = getProgramData<SphereGeoData>();

            Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());


            const Particle& particle = self.particleBufferPtr[primID];
            glm::vec3 P = ray.origin + ray.tmax * ray.direction;
            glm::vec3 N = glm::normalize(P - glm::vec3(particle.pos));

            glm::vec3 ffN = faceforward(N, -ray.direction, N);

            glm::vec3 geo_col = glm::vec3(self.globalColor);
            if (self.hasColorData) {
                geo_col = glm::vec3(self.colorBufferPtr[primID]);
            }

            if (prd.countEmitted)
                prd.emitted = geo_col * 0.2f;
            else
                prd.emitted = glm::vec3(0.0f);


            unsigned int seed = prd.seed;

            {
                const float z1 = rnd(seed);
                const float z2 = rnd(seed);

                glm::vec3 w_in;
                w_in = CosineSampleHemisphere(glm::vec2(z1, z2));
                Onb onb(N);
                onb.inverse_transform(w_in);
                prd.direction = w_in;
                prd.origin = P;

                prd.beta *= geo_col;
                prd.countEmitted = false;
            }

            const float z1 = rnd(seed);
            const float z2 = rnd(seed);


            // Calculate properties of light sample (for area based pdf)
            const float Ldist = length(prd.lpos - P);
            const glm::vec3 L = normalize(prd.lpos - P);
            const float nDl = dot(N, L);
            const float LnDl = -dot(prd.ldir, L);

            float weight = 0.0f;
            if (nDl > 0.0f && LnDl > 0.0f) {
                // const bool occluded = traceOcclusion(params.handle, P, L,
                //    0.01f,        // tmin
                //    Ldist - 0.01f // tmax
                //);
                float3 Pn = make_float3(P.x, P.y, P.z);
                float3 Ln = make_float3(L.x, L.y, L.z);
                unsigned int occluded = 0;
                optixTrace(prd.world, Pn, Ln, 0.01f, Ldist - 0.01f, 0.0f, (OptixVisibilityMask) -1,
                    OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, 1, 2, 1, occluded);

                if (!occluded) {
                    weight = nDl * LnDl / (MMO_PI * Ldist * Ldist);
                }
            }

            prd.radiance += glm::vec3(0.6f) * weight;
        }


        MM_OPTIX_CLOSESTHIT_KERNEL(sphere_closesthit_occlusion)() {
            /*PerRayData& prd = getPerRayData<PerRayData>();
            prd.inShadow = true;*/
            optixSetPayload_0(1);
        }


        MM_OPTIX_BOUNDS_KERNEL(sphere_bounds)(const void* geomData, box3f& primBounds, const unsigned int primID) {
            /*const SphereGeoData& self = *(const SphereGeoData*) geomData;

            const Particle& particle = self.particleBufferPtr[primID];*/
            Particle const* particles = (Particle const*) geomData;
            Particle const& particle = particles[primID];


            primBounds.lower = glm::vec3(particle.pos) - particle.pos.w;
            primBounds.upper = glm::vec3(particle.pos) + particle.pos.w;

            // printf("BOUNDS: %d with radius %f and box %f %f %f %f %f %f\n", primID, self.radius,
            // primBounds.lower.x, primBounds.lower.y, primBounds.lower.z, primBounds.upper.x, primBounds.upper.y,
            // primBounds.upper.z);
        }
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
