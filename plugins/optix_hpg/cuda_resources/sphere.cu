#include "hip/hip_runtime.h"
#include "perraydata.h"
#include "sphere.h"

#include "glm/glm.hpp"
#include "optix/utils_device.h"

#include "optix/random.h"

#include "glm/gtx/component_wise.hpp"

// OptiX SDK and rtxpkd

namespace megamol {
namespace optix_hpg {
    namespace device {
        inline __device__ void intersectSphere(const Particle& particle, const float particleRadius, const Ray& ray) {
            // Raytracing Gems Intersection Code (Chapter 7)
            const glm::vec3 pos = glm::vec3(particle.pos);
            const glm::vec3 oc = ray.origin - pos;
            const float sqrRad = particleRadius * particleRadius;

            // const float  a = dot(ray.direction, ray.direction);
            const float b = glm::dot(-oc, ray.direction);
            const glm::vec3 temp = oc + b * ray.direction;
            const float delta = sqrRad - glm::dot(temp, temp);

            if (delta < 0.0f)
                return;

            const float c = glm::dot(oc, oc) - sqrRad;
            const float q = b + copysignf(sqrtf(delta), b);

            {
                const float ta = c / q;
                const float tb = q;
                const float t = fminf(ta, tb);
                if (t < 0.f)
                    return;
                if (t > ray.tmin && t < ray.tmax) {
                    optixReportIntersection(t, 0);
                }
            }
        }

        MM_OPTIX_INTERSECTION_KERNEL(sphere_intersect)() {
            // printf("ISEC\n");

            const int primID = optixGetPrimitiveIndex();

            const auto& self = getProgramData<SphereGeoData>();

            auto const ray =
                Ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

            const Particle& particle = self.particleBufferPtr[primID];
            // float tmp_hit_t = ray.tmax;
            /*if (intersectSphere(particle, particle.pos.w, ray, tmp_hit_t)) {
                optixReportIntersection(tmp_hit_t, 0);
            }*/
            intersectSphere(particle, particle.pos.w, ray);
        }

        // OptiX SDK
        // Path tracer example

        //
        // Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
        //
        // Redistribution and use in source and binary forms, with or without
        // modification, are permitted provided that the following conditions
        // are met:
        //  * Redistributions of source code must retain the above copyright
        //    notice, this list of conditions and the following disclaimer.
        //  * Redistributions in binary form must reproduce the above copyright
        //    notice, this list of conditions and the following disclaimer in the
        //    documentation and/or other materials provided with the distribution.
        //  * Neither the name of NVIDIA CORPORATION nor the names of its
        //    contributors may be used to endorse or promote products derived
        //    from this software without specific prior written permission.
        //
        // THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
        // EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
        // IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
        // PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
        // CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
        // EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
        // PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
        // PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
        // OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
        // (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
        // OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
        //

        //
        // Modified 2021 MegaMol Dev Team
        //

        MM_OPTIX_CLOSESTHIT_KERNEL(sphere_closesthit)() {
            const int primID = optixGetPrimitiveIndex();
            PerRayData& prd = getPerRayData<PerRayData>();
            /*prd.primID = primID;
            prd.t = optixGetRayTmax();*/

            const auto& self = getProgramData<SphereGeoData>();

            Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());


            const Particle& particle = self.particleBufferPtr[primID];
            glm::vec3 P = ray.origin + ray.tmax * ray.direction;
            glm::vec3 N = glm::normalize(P - glm::vec3(particle.pos));

            glm::vec3 ffN = faceforward(N, -ray.direction, N);

            glm::vec3 geo_col = glm::vec3(self.globalColor);
            if (self.hasColorData) {
                geo_col = glm::vec3(self.colorBufferPtr[primID]);
            }

            set_depth(prd, ray.tmax);
            lighting(prd, geo_col, P, ffN);
        }


        MM_OPTIX_CLOSESTHIT_KERNEL(sphere_closesthit_occlusion)() {
            /*PerRayData& prd = getPerRayData<PerRayData>();
            prd.inShadow = true;*/
            optixSetPayload_0(1);
        }


        MM_OPTIX_BOUNDS_KERNEL(sphere_bounds)(const void* geomData, box3f& primBounds, const unsigned int primID) {
            /*const SphereGeoData& self = *(const SphereGeoData*) geomData;

            const Particle& particle = self.particleBufferPtr[primID];*/
            Particle const* particles = (Particle const*) geomData;
            Particle const& particle = particles[primID];


            primBounds.lower = glm::vec3(particle.pos) - particle.pos.w;
            primBounds.upper = glm::vec3(particle.pos) + particle.pos.w;

            // printf("BOUNDS: %d with radius %f and box %f %f %f %f %f %f\n", primID, self.radius,
            // primBounds.lower.x, primBounds.lower.y, primBounds.lower.z, primBounds.upper.x, primBounds.upper.y,
            // primBounds.upper.z);
        }
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
