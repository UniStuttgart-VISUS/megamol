#include "hip/hip_runtime.h"
#include "optix/utils_device.h"

#include "mesh.h"
#include "perraydata.h"

#include "optix/random.h"

#include "hip/hip_runtime.h"

namespace megamol {
namespace optix_hpg {
    namespace device {
        // OptiX SDK
        // Path tracer example

        //
        // Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
        //
        // Redistribution and use in source and binary forms, with or without
        // modification, are permitted provided that the following conditions
        // are met:
        //  * Redistributions of source code must retain the above copyright
        //    notice, this list of conditions and the following disclaimer.
        //  * Redistributions in binary form must reproduce the above copyright
        //    notice, this list of conditions and the following disclaimer in the
        //    documentation and/or other materials provided with the distribution.
        //  * Neither the name of NVIDIA CORPORATION nor the names of its
        //    contributors may be used to endorse or promote products derived
        //    from this software without specific prior written permission.
        //
        // THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
        // EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
        // IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
        // PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
        // CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
        // EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
        // PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
        // PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
        // OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
        // (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
        // OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
        //

        //
        // Modified 2021 MegaMol Dev Team
        //

        MM_OPTIX_CLOSESTHIT_KERNEL(mesh_closesthit)() {
            const int primID = optixGetPrimitiveIndex();
            PerRayData& prd = getPerRayData<PerRayData>();

            const auto& self = getProgramData<MeshGeoData>();

            const Ray ray(optixGetWorldRayOrigin(), optixGetWorldRayDirection(), optixGetRayTmin(), optixGetRayTmax());

            /*const float2 tmp_bary = optixGetTriangleBarycentrics();
            const glm::vec2 bary = glm::vec2(tmp_bary.x, tmp_bary.y);*/
            const glm::uvec3 indices = self.index_buffer[primID];
            const glm::vec3 v0 = self.vertex_buffer[indices.x];
            const glm::vec3 v1 = self.vertex_buffer[indices.y];
            const glm::vec3 v2 = self.vertex_buffer[indices.z];
            const glm::vec3 N = normalize(cross(v1 - v0, v2 - v0));

            // const auto tmp_N = optixTransformNormalFromObjectToWorldSpace(make_float3(normal.x, normal.y, normal.z));

            // const glm::vec3 N = normalize(glm::vec3(tmp_N.x, tmp_N.y, tmp_N.z));
            // const glm::vec3 N = normalize(normal);
            const glm::vec3 P = ray.origin + ray.tmax * ray.direction;

            glm::vec3 ffN = faceforward(N, -ray.direction, N);

            /*glm::vec3 geo_col = glm::vec3(self.globalColor);
            if (self.hasColorData) {
                geo_col = glm::vec3(self.colorBufferPtr[primID]);
            }*/

            glm::vec3 geo_col = glm::vec3(1.f, 1.f, 0.f);

            set_depth(prd, ray.tmax);
            lighting(prd, geo_col, P, ffN);
        }

        MM_OPTIX_CLOSESTHIT_KERNEL(mesh_closesthit_occlusion)() {
            optixSetPayload_0(1);
        }
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
