#include "optix/utils_device.h"

#include "miss.h"
#include "perraydata.h"

namespace megamol {
namespace optix_hpg {
    namespace device {
        MM_OPTIX_MISS_KERNEL(miss_program)() {
            PerRayData& prd = getPerRayData<PerRayData>();
            const auto& self = getProgramData<MissData>();

            prd.radiance = glm::vec3(self.bg);
            prd.done = true;
        }

        MM_OPTIX_MISS_KERNEL(miss_program_occlusion)() {}
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
