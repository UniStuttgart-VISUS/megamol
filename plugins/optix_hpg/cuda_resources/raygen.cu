#include "hip/hip_runtime.h"
#include "camera.h"
#include "raygen.h"

#include "optix/random.h"
#include "optix/utils_device.h"

namespace megamol {
namespace optix_hpg {
    namespace device {
        // OptiX SDK
        // Path tracer example

        //
        // Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
        //
        // Redistribution and use in source and binary forms, with or without
        // modification, are permitted provided that the following conditions
        // are met:
        //  * Redistributions of source code must retain the above copyright
        //    notice, this list of conditions and the following disclaimer.
        //  * Redistributions in binary form must reproduce the above copyright
        //    notice, this list of conditions and the following disclaimer in the
        //    documentation and/or other materials provided with the distribution.
        //  * Neither the name of NVIDIA CORPORATION nor the names of its
        //    contributors may be used to endorse or promote products derived
        //    from this software without specific prior written permission.
        //
        // THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
        // EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
        // IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
        // PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
        // CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
        // EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
        // PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
        // PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
        // OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
        // (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
        // OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
        //

        //
        // Modified 2021 MegaMol Dev Team
        //

        // code partially from: https://github.com/UniStuttgart-VISUS/rtxpkd_ldav2020
    // ======================================================================== //
    // Copyright 2018-2019 Ingo Wald                                            //
    //                                                                          //
    // Licensed under the Apache License, Version 2.0 (the "License");          //
    // you may not use this file except in compliance with the License.         //
    // You may obtain a copy of the License at                                  //
    //                                                                          //
    //     http://www.apache.org/licenses/LICENSE-2.0                           //
    //                                                                          //
    // Unless required by applicable law or agreed to in writing, software      //
    // distributed under the License is distributed on an "AS IS" BASIS,        //
    // WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
    // See the License for the specific language governing permissions and      //
    // limitations under the License.                                           //
    // ======================================================================== //

    // ======================================================================== //
    // Modified 2019-2020 VISUS - University of Stuttgart                       //
    // ======================================================================== //



//#define RANDVEC3F glm::vec3(rnd(42), rnd(42), rnd(42))
#define RANDVEC3F glm::vec3(rnd(seed), rnd(seed), rnd(seed))

    inline __device__ glm::vec3 random_in_unit_sphere(unsigned int& seed) {
        glm::vec3 p;
        do {
            p = 2.0f * RANDVEC3F - glm::vec3(1, 1, 1);
        } while (glm::dot(p, p) >= 1.0f);
        return p;
    }

        inline __device__ glm::vec4 traceRay(
            const RayGenData& self, Ray& ray, unsigned int& seed /*, Random& rnd*/, PerRayData& prd, glm::vec4& bg, int maxBounces) {

            unsigned int p0 = 0;
            unsigned int p1 = 0;
            packPointer(&prd, p0, p1);

            glm::vec3 col(1.f);


            #if 0
            for (;;) {
                prd.wo = -ray.direction;
                optixTrace(self.world, (const float3&) ray.origin, (const float3&) ray.direction, ray.tmin, ray.tmax, 0,
                    (OptixVisibilityMask) -1,
                    /*rayFlags     */ OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    /*SBToffset    */ 0,
                    /*SBTstride    */ 2,
                    /*missSBTIndex */ 0, p0, p1);


                /*if (prd.depth > 0) {
                    col += prd.attenuation * prd.result;
                } else {
                    col += prd.result;
                }*/

                // col += prd.emitted;
                col += prd.radiance * prd.beta;

                if (prd.done || prd.depth >= maxBounces)
                    break;

                ++prd.depth;

                ray.origin = prd.origin;
                ray.direction = prd.direction;
            }
            col += prd.emitted;
            return glm::vec4(col.x, col.y, col.z, 1.0f);
            //return glm::vec4(1, 1, 1, 1.0f);
            // return glm::vec4(prd.radiance, 1.0f);
            #else
            for (int depth = 0;true;++depth) {
                prd.particleID = -1;

                optixTrace(self.world, (const float3&)ray.origin, (const float3&)ray.direction, ray.tmin, ray.tmax, 0,
                    (OptixVisibilityMask)-1,
                    /*rayFlags     */ OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    /*SBToffset    */ 0,
                    /*SBTstride    */ 2,
                    /*missSBTIndex */ 0, p0, p1);
                if (prd.particleID == -1) {
                    return glm::vec4(col * glm::vec3(0.8f), 1.0f);
                }

                glm::vec3 N = (ray.origin + prd.t * ray.direction) - prd.pos;
                if (glm::dot(N, ray.direction) > 0.f)
                    N = -N;
                N = glm::normalize(N);

                if (maxBounces == 0) {
                    return glm::vec4(prd.albedo * (.2f + .6f * fabsf(glm::dot(N, ray.direction))), 1.0f);
                }

                col *= prd.albedo;

                if (depth >= maxBounces)
                    return glm::vec4(0.1f, 0.1f, 0.1f, 1.0f);

                auto scattered_origin = ray.origin + prd.t * ray.direction;
                auto scattered_direction = N + random_in_unit_sphere(seed);
                ray = Ray(/* origin   : */ scattered_origin,
                    /* direction: */ glm::normalize(scattered_direction),
                    /* tmin     : */ 1e-3f,
                    /* tmax     : */ 1e+8f);
            }
            #endif
        }

        MM_OPTIX_RAYGEN_KERNEL(raygen_program)() {
            // printf("RAYGEN1\n");
            const RayGenData& self = getProgramData<RayGenData>();
            auto const index = optixGetLaunchIndex();
            glm::ivec2 pixelID = glm::ivec2(index.x, index.y);

            if (pixelID.x >= self.fbSize.x)
                return;
            if (pixelID.y >= self.fbSize.y)
                return;
            //const int pixelIdx = pixelID.x + self.fbSize.x * pixelID.y;

            const FrameState* fs = &self.frameStateBuffer[0];

            /*auto frame_idx = self.colorBufferPtr[pixelIdx].w;
            if (fs->changed) {
                frame_idx = 0.0f;
                self.colorBufferPtr[pixelIdx].w = 0.0f;
            }*/
            // auto const old_col = self.colorBufferPtr[pixelIdx];
            float4 old_col;
            surf2Dread(&old_col, self.col_surf, pixelID.x * sizeof(float4), pixelID.y, hipBoundaryModeZero);

            unsigned int seed = tea<16>(pixelID.y * self.fbSize.x + pixelID.x, fs->frameIdx);


            glm::vec4 col(0.f);
            glm::vec4 bg = fs->background;

            // printf("RAYGEN FS %f\n", fs->near);

            auto i = fs->samplesPerPixel;

            float depth = FLT_MAX;

            #if 0
            do {
                PerRayData prd;

                prd.depth = 0;

                prd.radiance = glm::vec3(0.f);
                prd.pdf = 1.0f;

                prd.countDepth = true;
                prd.ray_depth = FLT_MAX;

                prd.beta = glm::vec3(1.f);

                prd.seed = seed;
                prd.done = false;

                prd.world = self.world;

                prd.countEmitted = true;
                prd.emitted = glm::vec3(0.f);

                prd.intensity = fs->intensity;

                // Random rnd(pixelIdx, 0);

                float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x + rnd(seed)) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y + rnd(seed)) / self.fbSize.y);
                /*float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y) / self.fbSize.y);*/
                auto ray = generateRay(*fs, u, v);

                prd.origin = ray.origin;
                prd.direction = ray.direction;

                prd.lpos = ray.origin;
                prd.ldir = fs->camera_front;

                col += traceRay(self, ray /*, rnd*/, prd, bg, fs->maxBounces);

                depth = fminf(depth, prd.ray_depth);
            } while (--i);
            #else
            PerRayData prd;
            do {
                prd.countDepth = true;
                prd.ray_depth = FLT_MAX;
                float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x + rnd(seed)) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y + rnd(seed)) / self.fbSize.y);
                /*float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y) / self.fbSize.y);*/
                auto ray = generateRay(*fs, u, v);
                col += traceRay(self, ray, seed /*, rnd*/, prd, bg, fs->maxBounces);
                depth = fminf(depth, prd.ray_depth);
            } while (--i);
            #endif
            col /= (float) fs->samplesPerPixel;
            // col.w = frame_idx + 1;
            //++col.w;

            if (fs->frameIdx > 0) {
                const float a = 1.0f / static_cast<float>(fs->frameIdx + 1);
                col = lerp(glm::vec4(static_cast<float>(old_col.x), static_cast<float>(old_col.y),
                               static_cast<float>(old_col.z), static_cast<float>(old_col.w)),
                    col, a);
                // col.w = frame_idx + 1;
            }

            if (depth < FLT_MAX) {
                depth = (fs->depth_params.z / depth) - (fs->depth_params.x);
                depth = 0.5f * (depth + 1.0f);
            } else {
                depth = 1.f;
                col = bg;
            }
            surf2Dwrite(depth, self.depth_surf, pixelID.x * sizeof(float), pixelID.y, hipBoundaryModeZero);

            surf2Dwrite(make_float4(col.r, col.g, col.b, col.a), self.col_surf, pixelID.x * sizeof(float4), pixelID.y,
                hipBoundaryModeZero);
            /*surf2Dwrite(make_float4(1, 1, 1, 1), self.col_surf, pixelID.x * sizeof(float4), pixelID.y,
                hipBoundaryModeZero);*/

            
            //surf2Dwrite(1, self.depth_surf, pixelID.x * sizeof(float), pixelID.y, hipBoundaryModeZero);
        }
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
