#include "hip/hip_runtime.h"
#include "camera.h"
#include "raygen.h"

#include "optix/random.h"
#include "optix/utils_device.h"

namespace megamol {
namespace optix_hpg {
    namespace device {
        // OptiX SDK
        // Path tracer example

        //
        // Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
        //
        // Redistribution and use in source and binary forms, with or without
        // modification, are permitted provided that the following conditions
        // are met:
        //  * Redistributions of source code must retain the above copyright
        //    notice, this list of conditions and the following disclaimer.
        //  * Redistributions in binary form must reproduce the above copyright
        //    notice, this list of conditions and the following disclaimer in the
        //    documentation and/or other materials provided with the distribution.
        //  * Neither the name of NVIDIA CORPORATION nor the names of its
        //    contributors may be used to endorse or promote products derived
        //    from this software without specific prior written permission.
        //
        // THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
        // EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
        // IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
        // PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
        // CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
        // EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
        // PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
        // PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
        // OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
        // (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
        // OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
        //

        //
        // Modified 2021 MegaMol Dev Team
        //

        inline __device__ glm::vec4 traceRay(
            const RayGenData& self, Ray& ray /*, Random& rnd*/, PerRayData& prd, glm::vec4& bg, int maxBounces) {

            unsigned int p0 = 0;
            unsigned int p1 = 0;
            packPointer(&prd, p0, p1);

            glm::vec3 col(0.f);

            for (;;) {
                prd.wo = -ray.direction;
                optixTrace(self.world, (const float3&) ray.origin, (const float3&) ray.direction, ray.tmin, ray.tmax, 0,
                    (OptixVisibilityMask) -1,
                    /*rayFlags     */ OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                    /*SBToffset    */ 0,
                    /*SBTstride    */ 2,
                    /*missSBTIndex */ 0, p0, p1);


                /*if (prd.depth > 0) {
                    col += prd.attenuation * prd.result;
                } else {
                    col += prd.result;
                }*/

                // col += prd.emitted;
                col += prd.radiance * prd.beta;

                if (prd.done || prd.depth >= maxBounces)
                    break;

                ++prd.depth;

                ray.origin = prd.origin;
                ray.direction = prd.direction;
            }
            col += prd.emitted;
            return glm::vec4(col.x, col.y, col.z, 1.0f);
            // return glm::vec4(prd.radiance, 1.0f);
        }

        MM_OPTIX_RAYGEN_KERNEL(raygen_program)() {
            // printf("RAYGEN1\n");
            const RayGenData& self = getProgramData<RayGenData>();
            auto const index = optixGetLaunchIndex();
            glm::ivec2 pixelID = glm::ivec2(index.x, index.y);

            if (pixelID.x >= self.fbSize.x)
                return;
            if (pixelID.y >= self.fbSize.y)
                return;
            const int pixelIdx = pixelID.x + self.fbSize.x * pixelID.y;

            const FrameState* fs = &self.frameStateBuffer[0];

            /*auto frame_idx = self.colorBufferPtr[pixelIdx].w;
            if (fs->changed) {
                frame_idx = 0.0f;
                self.colorBufferPtr[pixelIdx].w = 0.0f;
            }*/
            // auto const old_col = self.colorBufferPtr[pixelIdx];
            float4 old_col;
            surf2Dread(&old_col, self.col_surf, pixelID.x * sizeof(float4), pixelID.y, hipBoundaryModeZero);

            unsigned int seed = tea<16>(pixelID.y * self.fbSize.x + pixelID.x, fs->frameIdx);


            glm::vec4 col(0.f);
            glm::vec4 bg = fs->background;

            // printf("RAYGEN FS %f\n", fs->near);

            auto i = fs->samplesPerPixel;

            float depth = FLT_MAX;

            do {
                PerRayData prd;

                prd.depth = 0;

                prd.radiance = glm::vec3(0.f);
                prd.pdf = 1.0f;

                prd.countDepth = true;
                prd.ray_depth = FLT_MAX;

                prd.beta = glm::vec3(1.f);

                prd.seed = seed;
                prd.done = false;

                prd.world = self.world;

                prd.countEmitted = true;
                prd.emitted = glm::vec3(0.f);

                prd.intensity = fs->intensity;

                // Random rnd(pixelIdx, 0);

                float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x + rnd(seed)) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y + rnd(seed)) / self.fbSize.y);
                /*float u = -fs->rw + (fs->rw + fs->rw) * float(pixelID.x) / self.fbSize.x;
                float v = -(fs->th + (-fs->th - fs->th) * float(pixelID.y) / self.fbSize.y);*/
                auto ray = generateRay(*fs, u, v);

                prd.origin = ray.origin;
                prd.direction = ray.direction;

                prd.lpos = ray.origin;
                prd.ldir = fs->camera_front;

                col += traceRay(self, ray /*, rnd*/, prd, bg, fs->maxBounces);

                depth = fminf(depth, prd.ray_depth);
            } while (--i);
            col /= (float) fs->samplesPerPixel;
            // col.w = frame_idx + 1;
            //++col.w;

            if (fs->frameIdx > 0) {
                const float a = 1.0f / static_cast<float>(fs->frameIdx + 1);
                col = lerp(glm::vec4(static_cast<float>(old_col.x), static_cast<float>(old_col.y),
                               static_cast<float>(old_col.z), static_cast<float>(old_col.w)),
                    col, a);
                // col.w = frame_idx + 1;
            }

            surf2Dwrite(make_float4(col.r, col.g, col.b, col.a), self.col_surf, pixelID.x * sizeof(float4), pixelID.y,
                hipBoundaryModeZero);

            if (depth < FLT_MAX) {
                depth = (fs->depth_params.z / depth) - (fs->depth_params.x);
                depth = 0.5f * (depth + 1.0f);
            } else {
                depth = 1.f;
            }
            surf2Dwrite(depth, self.depth_surf, pixelID.x * sizeof(float), pixelID.y, hipBoundaryModeZero);
        }
    } // namespace device
} // namespace optix_hpg
} // namespace megamol
