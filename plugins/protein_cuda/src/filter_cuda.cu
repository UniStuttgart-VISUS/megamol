#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "filter_cuda.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"

// Parameters in constant memory
__constant__ FilterParams fparams;


__device__ float getDistanceSq(float3 posA, float3 posB) {
    
    return (posA.x-posB.x)*(posA.x-posB.x) + (posA.y-posB.y)*(posA.y-posB.y) +
           (posA.z-posB.z)*(posA.z-posB.z); 
}


/*
 * hashValMultiGrid
 * 
 * Hash function resulting in a distribution of values with eight successive
 * hash values remaining inside the same cube.
 * 
 */
__device__
unsigned int hashValMultiGrid(int3 gridPos) {
    
    // 
    // (1) Grid size of coarse grid: 
    //      coarseGridSize  = fparams.gridSize.x / 2
    //
    // (2) Address of subcube in coarse grid: 
    //      subCubeAddr = 
    //      (int)(gridPos.y/2) * coarseGridSize 
    //      + (int)(gridPos.x/2) 
    //      + (int)(gridPos.z/2) * coarseGridSize  * coarseGridSize ;
    // 
    // (3) Hash value within one subcube: 
    //      h = ((gridPos.x)%2 + gridPos.y%2 * 2 + 2 * 2 * (gridPos.z%2))
    // 
    // (4) Actual hash value: 
    //      h + 8 * subCubeAddr
    
    unsigned int coarseGridSize = fparams.gridSize.x / 2;
    
    // TODO: gridPos.x % 2 = gridPos.x & (2 - 1) ?
    return (gridPos.x % 2) + (((gridPos.y % 2) + (gridPos.z % 2) * 2)
            + 2 * 2 * (gridPos.y/2 * coarseGridSize + gridPos.x/2 
                    + gridPos.z/2 * coarseGridSize) * coarseGridSize) * 2;
}


/*
 * calcFilterHashGridD
 */
__global__
void calcFilterHashGridD(unsigned int *gridHash,  
                         unsigned int *gridIndex,
                         float3       *atmPos) {
                             
    unsigned int idx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    
    if(idx >= fparams.atmCntProt) 
        return;
    
    int3 gridPos = make_int3(floor((atmPos[idx].x - fparams.worldOrigin.x) / fparams.cellSize.x),
                             floor((atmPos[idx].y - fparams.worldOrigin.y) / fparams.cellSize.y),
                             floor((atmPos[idx].z - fparams.worldOrigin.z) / fparams.cellSize.z));

    // Wrap grid, assumes size is power of 2
    /*gridPos.x = gridPos.x & (fparams.gridSize.x - 1);  
    gridPos.y = gridPos.y & (fparams.gridSize.y - 1);
    gridPos.z = gridPos.z & (fparams.gridSize.z - 1);*/

    // Calculate hash value
    gridHash[idx]  = __umul24(__umul24(gridPos.z, fparams.gridSize.y), 
        fparams.gridSize.x) + __umul24(gridPos.y, fparams.gridSize.x) + gridPos.x;
    

    // Init index array 
    gridIndex[idx] = idx;
}



/*
 * reorderFilterDataD
 */
__global__
void reorderFilterDataD(unsigned int *cellStart,        
                        unsigned int *cellEnd,      
                        unsigned int *gridHash, 
                        unsigned int *gridIndex,
                        float3       *atmPos,
                        float3       *atmPosSorted) {
                        
    extern __shared__ unsigned int sharedHash[];    // blockSize + 1 elements
    
    unsigned int idx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int hash;

    if(idx < fparams.atmCntProt) {
        hash = gridHash[idx];

        // Load hash data into shared memory so that we can look 
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if(idx > 0 && threadIdx.x == 0) {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridHash[idx - 1];
        }
    }

    __syncthreads();
    
    if(idx < fparams.atmCntProt) {
        
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell
        
        if(idx == 0 || hash != sharedHash[threadIdx.x]) {
            cellStart[hash] = idx;
            if(idx > 0)
                cellEnd[sharedHash[threadIdx.x]] = idx;
        }

        if(idx == fparams.atmCntProt - 1) {
            cellEnd[hash] = idx + 1;
        }

        // Now use the sorted index to reorder the pos data
        atmPosSorted[idx] = atmPos[gridIndex[idx]];
        
        // macro does either global read or texture fetch
        //float4 pos = FETCH( oldPos, sortedIndex);       
    }
}



/*
 * calcSolventVisibilityD
 */
__global__
void calcSolventVisibilityD(unsigned int *cellStart,
                            unsigned int *cellEnd,
                            float3       *atmPos,
                            float3       *atmPosProtSorted,
                            bool         *isSolventAtom,
                            int          *atomVisibility) {

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx >= fparams.atmCnt) {
        return;
    }
    
    // Non-solvent atoms are visible
    if(!isSolventAtom[idx]) {
        atomVisibility[idx] = 0;
        return;
    }
    
    // Note: startIndex/endIndex are referring to the position in the sorted array
    unsigned int startIdx, endIdx, hash, at, x, y, z;
    
    // Get position of the atom
    float3 p = atmPos[idx];

    int3 gridPos = make_int3(floor((p.x - fparams.worldOrigin.x) / fparams.cellSize.x),
                             floor((p.y - fparams.worldOrigin.y) / fparams.cellSize.y),
                             floor((p.z - fparams.worldOrigin.z) / fparams.cellSize.z));
                             
    int3 discRangeCeil = make_int3(
        (gridPos.x + fparams.discRange.x < fparams.gridSize.x) ? 
            (gridPos.x + fparams.discRange.x) : fparams.gridSize.x,
        (gridPos.y + fparams.discRange.y < fparams.gridSize.y) ? 
            (gridPos.y + fparams.discRange.y) : fparams.gridSize.y,
        (gridPos.z + fparams.discRange.z < fparams.gridSize.z) ? 
            (gridPos.z + fparams.discRange.z) : fparams.gridSize.z);
        
    int3 discRangeFloor = make_int3(
        (gridPos.x - fparams.discRange.x >= 0) ? (gridPos.x - fparams.discRange.x) : 0,
        (gridPos.y - fparams.discRange.y >= 0) ? (gridPos.y - fparams.discRange.y) : 0,
        (gridPos.z - fparams.discRange.z >= 0) ? (gridPos.z - fparams.discRange.z) : 0);
    
    // Examine neighbouring cells within the given range
    for(z = discRangeFloor.z; z < discRangeCeil.z; z++) {
        for(y = discRangeFloor.y; y < discRangeCeil.y; y++) {
            for(x = discRangeFloor.x; x < discRangeCeil.x; x++) {
                    
                hash = __umul24(__umul24(z, fparams.gridSize.y), fparams.gridSize.x) + 
                       __umul24(y, fparams.gridSize.x) + 
                       x;
                        
                startIdx = cellStart[hash];
                
                if(startIdx == 0xffffffff) {
                   continue; // Cell is empty - continue with next cell
                }
                else {

                    endIdx = cellEnd[hash];
            
                    // Iterate over all atoms in this cell
                    for(at = startIdx; at < endIdx; at++) {
                        if(getDistanceSq(atmPosProtSorted[at], p) <= fparams.solvRangeSq) {
                            
                            atomVisibility[idx] = 1;
                            return; 
                        }
                    }
                }
            }
        }
    }
}



/*
 * calcSolventVisibilityAltD
 */
__global__
void calcSolventVisibilityAltD(unsigned int *cellStart,
                               unsigned int *cellEnd,
                               float3       *atmPos,
                               float3       *atmPosProtSorted,
                               bool         *isSolventAtom,
                               int          *atomVisibility,
                               int3          *neighbourCellPos) {
                                
    extern __shared__ bool sharedVisibility[];

    unsigned int atmIdx = blockIdx.x;
    unsigned int neighbourCellIdx = threadIdx.x;
    
    if(!((atmIdx >= fparams.atmCnt) || (neighbourCellIdx >= fparams.numNeighbours))) {
        
        sharedVisibility[threadIdx.x] = false;
         
        if(!isSolventAtom[atmIdx]) { // Non-solvent atoms are visible
            sharedVisibility[threadIdx.x] = true;            
        }
        else {        
            // Get position of the atom
            float3 p = atmPos[atmIdx];
        
            int3 gridPos = make_int3(floor((p.x - fparams.worldOrigin.x) / fparams.cellSize.x),
                                     floor((p.y - fparams.worldOrigin.y) / fparams.cellSize.y),
                                     floor((p.z - fparams.worldOrigin.z) / fparams.cellSize.z));
                                     
            
            int3 neighbourPos = make_int3((gridPos.x + neighbourCellPos[neighbourCellIdx].x) & (fparams.gridSize.x - 1),  // wrap grid
                                         (gridPos.y + neighbourCellPos[neighbourCellIdx].y) & (fparams.gridSize.y - 1),
                                         (gridPos.z + neighbourCellPos[neighbourCellIdx].z) & (fparams.gridSize.z - 1));
            
                                    
            unsigned int hash = __umul24(__umul24(neighbourPos.z, fparams.gridSize.y), fparams.gridSize.x) + 
                                __umul24(neighbourPos.y, fparams.gridSize.x) + 
                                neighbourPos.x;
                    
            unsigned int startIdx = cellStart[hash];
            
            if(startIdx != 0xffffffff) {
        
                unsigned int endIdx = cellEnd[hash];

        
                // Iterate over all atoms in the neighbour cell
                // TODO
                for(unsigned int at = startIdx; at < endIdx; at++) {
                    if(getDistanceSq(atmPosProtSorted[at], p) <= fparams.solvRangeSq) {
                        sharedVisibility[threadIdx.x] = true;
                        break;
                    }
                }
            }
        }
    }
    else
        return;
    
    __syncthreads();
    
    // Merge visibility information
    if(threadIdx.x == 0) {
        atomVisibility[atmIdx] = 0;
        for(int i = 0; i < fparams.numNeighbours; i++) {
            if(sharedVisibility[i] == true) {
                atomVisibility[atmIdx] = 1;
                break;
            }
        }
    }
}




extern "C" {
 
    
    /*
     * setFilterParams
     */
    void setFilterParams(FilterParams *hostParams) {
        // Copy parameters to constant memory
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fparams), hostParams, sizeof(FilterParams)));
    }


    /*
     * calcFilterHashGrid
     */
    void calcFilterHashGrid(unsigned int *gridHash,
                            unsigned int *gridIndex,
                            float        *atmPosProt,
                            unsigned int  atmCntProt) {
        
        // Compute grid size
        unsigned int numThreads = min(256, atmCntProt);
        unsigned int numBlocks  = (unsigned int)ceil((float)atmCntProt/(float)numThreads);
    
        // Execute the kernel
        calcFilterHashGridD <<< numBlocks, numThreads >>> (gridHash,
                                                           gridIndex,
                                                           (float3*) atmPosProt);
        
        getLastCudaError("calcFilterHashGridD");
    }


    /*
     * reorderFilterData
     */
    void reorderFilterData(unsigned int *cellStart,
                           unsigned int *cellEnd,
                           unsigned int *gridHash,
                           unsigned int *gridIndex,
                           float        *atmPosProt,
                           float        *atmPosProtSorted,
                           unsigned int  atmCntProt) {
  
        unsigned int numThreads = min(256, atmCntProt);
        unsigned int numBlocks  = (unsigned int)ceil((float)atmCntProt/(float)numThreads);
        
        // Compute memory size
        unsigned int memSize = sizeof(unsigned int)*(numThreads+1);
        
        // Execute kernel
        reorderFilterDataD <<< numBlocks, numThreads, memSize >>> (cellStart,
                                                                   cellEnd,
                                                                   gridHash,
                                                                   gridIndex,
                                                                   (float3*) atmPosProt,
                                                                   (float3*) atmPosProtSorted);
            
        getLastCudaError("reorderFilterDataD");
    }
                                       
    
    /*
     * calcSolventVisibilityAlt
     */
    void calcSolventVisibilityAlt(unsigned int *cellStart,
                                  unsigned int *cellEnd,
                                  float        *atmPos,
                                  float        *atmPosProtSorted,
                                  bool         *isSolventAtom,
                                  int          *atomVisibility,
                                  int          *neighbourCellPos,
                                  unsigned int  atmCnt,
                                  unsigned int  numNeighbours) {
                                   
        
        unsigned int numThreads = numNeighbours; // TODO this only works if number of neighbour cells is less than 512
        unsigned int memSize = numNeighbours*sizeof(bool);
        unsigned int numBlocks = atmCnt;
                                         
        // Execute kernel
        calcSolventVisibilityAltD <<< numBlocks, numThreads, memSize >>> (cellStart,
                                                          cellEnd,
                                                          (float3*) atmPos,
                                                          (float3*) atmPosProtSorted,
                                                          isSolventAtom,
                                                          atomVisibility,
                                                          (int3*) neighbourCellPos);
        
        getLastCudaError("calcSolventVisibilityAltD");
        
    }
    
    
    /*
     * calcSolventVisibility
     */
    void calcSolventVisibility(unsigned int *cellStart,
                               unsigned int *cellEnd,
                               float        *atmPos,
                               float        *atmPosProtSorted,
                               bool         *isSolventAtom,
                               int          *atomVisibility,
                               unsigned int  atmCnt) {
    
        // Compute grid size
        unsigned int numThreads = min(256, atmCnt);
        unsigned int numBlocks  = (unsigned int)ceil((float)atmCnt/(float)numThreads);                                     
                                         
        // Execute kernel
        calcSolventVisibilityD <<< numBlocks, numThreads >>> (cellStart,
                                                              cellEnd,
                                                              (float3*) atmPos,
                                                              (float3*) atmPosProtSorted,
                                                              isSolventAtom,
                                                              atomVisibility);
        
        getLastCudaError("calcSolventVisibilityD");                                                              
    }

                                 

} // extern "C"

