#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// Simple 3D volume renderer

#ifndef _CUDARAYCASTER_KERNEL_CU_
#define _CUDARAYCASTER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <iostream>
#include <omp.h>
#include <fstream>
#include <vector>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

#define CYLINDER

typedef unsigned int  uint;
typedef unsigned char uchar;

hipEvent_t evtStart, evtStop;

hipArray *d_volumeArray = 0;
hipArray *d_customTransferFuncArray;

//hipArray *d_isoValArray = 0;

__device__ int d_numIsoVals = 0;
__device__ float d_iso1 = -1.0f;
__device__ float d_iso2 = -1.0f;
__device__ float d_iso3 = -1.0f;
__device__ float d_iso4 = -1.0f;

std::vector<float> fpsVec;

typedef float VolumeType;

texture<VolumeType, 3, hipReadModeElementType> tex;
texture<float4, 1, hipReadModeElementType> customTransferTex;
float minVal, maxVal;

typedef struct {
	float4 m[4];
} mymatrix;

__constant__ mymatrix c_invViewMatrix;  // inverse view matrix
__constant__ mymatrix c_mvpMatrix; // modelview-projection-matrix

struct Ray {
	float3 o;   // origin
	float3 d;   // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar) {
	// compute intersection of ray with all six bbox planes
	float3 invR = make_float3(1.0f) / r.d;
	float3 tbot = invR * (boxmin - r.o);
	float3 ttop = invR * (boxmax - r.o);

	// re-order intersections to find smallest and largest on each axis
	float3 tmin = fminf(ttop, tbot);
	float3 tmax = fmaxf(ttop, tbot);

	// find the largest tmin and the smallest tmax
	float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
	float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix with translation
__device__ float4 mul(const mymatrix &M, const float4 &v) {
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = dot(v, M.m[3]);
	return r;
}

// transform vector by matrix (no translation)
__device__ float3 mul(const mymatrix &M, const float3 &v) {
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

__device__ uint rgbaFloatToInt(float4 rgba) {
	rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = __saturatef(rgba.w);
	return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) | (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__device__ float4 performLighting(float3 normal, float3 camDirection, float3 lightDirection, float4 surfaceColor, float4 lightParams) {
	float3 lightDir = normalize(lightDirection);
	float3 n = normalize(normal);
	float3 dir = normalize(camDirection);
	float ndotl = dot(n, lightDir);

	float3 r = normalize(2.0f * ndotl * n - lightDir);
	float spec = powf(max(dot(r, -dir), 0.0), lightParams.w);
	float diff = max(abs(ndotl), 0.0);

	float3 col = make_float3(surfaceColor.x, surfaceColor.y, surfaceColor.z);
	float3 result = lightParams.x * col + lightParams.y * diff * col + lightParams.z * spec * make_float3(1, 1, 1);

	return make_float4(result, surfaceColor.w);
}

__device__ bool pointBehindPlane(float3 point, float4 plane) {
	float dist = plane.x * point.x + plane.y * point.y + plane.z * point.z + plane.w;
	return (dist < 0);
}

__global__ void
d_render(uint *d_output, float *d_depth_output, uint imageW, uint imageH, float fovx, float fovy, float3 camPos, float3 camDir, float3 camUp, float3 camRight, float zNear,
		float density, float brightness, float transferOffset, float transferScale, float minVal, float maxVal,
		const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f), const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f), hipExtent volSize = make_hipExtent(1, 1, 1),
		const float3 lightDir = make_float3(1.0f, 1.0f, 1.0f), const float4 lightParams = make_float4(0.3f, 0.5f, 0.4f, 10.0f),
		const float4 plane = make_float4(0.0f, 0.0f, 0.0f, 0.0), bool enablePlane = false) {

	const int maxSteps = 1000;
	//const float tstep = 0.0009765625f;

	const float isoVals[4] = { d_iso1, d_iso2, d_iso3, d_iso4 };

	//const float tstep = (boxMax.x - boxMin.x) / (float)maxSteps;
	const float tstep = length(boxMax - boxMin) / (float)maxSteps;
	const float opacityThreshold = 0.95f;

	uint x = blockIdx.x*blockDim.x + threadIdx.x;
	uint y = blockIdx.y*blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;
	
	float u = (x / (float)imageW)*2.0f - 1.0f;
	float v = (y / (float)imageH)*2.0f - 1.0f;

	float3 oL = (tan(fovx * 0.5f) * zNear) * (-camRight) + (tan(fovy * 0.5) * zNear) * camUp + camDir * zNear + camPos;
	float3 uL = (tan(fovx * 0.5f) * zNear) * (-camRight) + (tan(fovy * 0.5) * zNear) * (-camUp) + camDir * zNear + camPos;
	float3 oR = (tan(fovx * 0.5f) * zNear) * camRight + (tan(fovy * 0.5) * zNear) * camUp + camDir * zNear + camPos;
	float3 uR = (tan(fovx * 0.5f) * zNear) * camRight + (tan(fovy * 0.5) * zNear) * (-camUp) + camDir * zNear + camPos;

	float3 targetL = lerp(uL, oL, (v + 1.0f) * 0.5f);
	float3 targetR = lerp(uR, oR, (v + 1.0f) * 0.5f);

	float3 target = lerp(targetL, targetR, (u + 1.0f) * 0.5f);

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = camPos;
	eyeRay.d = normalize(target - camPos);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

	if (!hit) {
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(0.0f));
		d_depth_output[y*imageW + x] = 1.0f;
		return;
	} 
	/*else {
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(1.0f));
		return;
	}*/

	if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

	// march along ray from front to back, accumulating color
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d*tnear;
	float3 step = eyeRay.d*tstep;

	float3 diff = boxMax - boxMin;

	if (d_numIsoVals < 1) { // no isosurfaces to render
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(0.0f));
		return;
	}

	if (d_numIsoVals > 0 && d_iso1 < 0.0f) {
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(0.0f));
		return;
	}

	/*if (d_numIsoVals > 0) {
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(d_iso1));
		return;
	}*/

	float3 sP;
	sP.x = (pos.x - boxMin.x) / diff.x;
	sP.y = (pos.y - boxMin.y) / diff.y;
	sP.z = (pos.z - boxMin.z) / diff.z;
	float val = (tex3D(tex, sP.x, sP.y, sP.z) - minVal) / (maxVal - minVal);

	float isoDiff = 0;
	float isoDiffOld = val - isoVals[0];

	float isoDiffs[4] = { 0, 0, 0, 0 };
	float isoDiffsOld[4] = { val - isoVals[0], val - isoVals[1], val - isoVals[2], val - isoVals[3] };

	float3 voxelSize = make_float3(1.0f / (float)volSize.width, 1.0f / (float)volSize.height, 1.0f / (float)volSize.depth);

	float alpha = 1.0f / (float)d_numIsoVals;
	// TODO change colors
	float4 colors[4] = { make_float4(1.0f, 0.0f, 0.0f, alpha),
							make_float4(0.0f, 1.0f, 0.0f, alpha),
							make_float4(0.0f, 0.0f, 1.0f, alpha),
							make_float4(1.0f, 0.0f, 0.0f, alpha) };

	bool firstHit = true;
	float3 firstHitPos;

	for (int i = 0; i<maxSteps; i++) {
		// read from 3D texture
		// remap position to [0, 1] coordinates
		//float sample = tex3D(tex, pos.x*0.5f + 0.5f, pos.y*0.5f + 0.5f, pos.z*0.5f + 0.5f);
		float3 samplePos;
		samplePos.x = (pos.x - boxMin.x) / diff.x;
		samplePos.y = (pos.y - boxMin.y) / diff.y;
		samplePos.z = (pos.z - boxMin.z) / diff.z;

		float curDist = 0.0f;
#ifdef CYLINDER
		curDist = sqrt((0.5f - samplePos.x) * (0.5f - samplePos.x) + (0.5f - samplePos.y) * (0.5f - samplePos.y));
		if (curDist > 0.5) {
			float2 middir = make_float2(0.5f, 0.5f) - make_float2(samplePos.x, samplePos.y);
			middir = normalize(middir);
			samplePos.x = samplePos.x + (curDist - 0.5f) * middir.x;
			samplePos.y = samplePos.y + (curDist - 0.5f) * middir.y;
		}
#endif CYLINDER

		float sample = (tex3D(tex, samplePos.x, samplePos.y, samplePos.z) - minVal) / (maxVal - minVal);

		if (pointBehindPlane(samplePos, plane) || curDist > 0.5f) {

			for (int isoIndex = 0; isoIndex < d_numIsoVals; isoIndex++) {
				isoDiffs[isoIndex] = sample - isoVals[isoIndex];
				isoDiffsOld[isoIndex] = isoDiffs[isoIndex];
			}

			t += tstep;
			if (t > tfar) break;
			pos += step;
			continue;
		}

		for (int isoIndex = 0; isoIndex < d_numIsoVals; isoIndex++) {

			isoDiffs[isoIndex] = sample - isoVals[isoIndex];

			if ((isoDiffs[isoIndex] * isoDiffsOld[isoIndex]) <= 0.0f) {

				// interpolated exact position of the isosurface
				float3 isoPos = lerp(pos - step, pos, isoDiffsOld[isoIndex] / (isoDiffsOld[isoIndex] - isoDiffs[isoIndex]));

				// if this is the first isosurface hit for this ray, remember the surface point.
				if (firstHit) {
					firstHitPos = isoPos;
					firstHit = false;
				}

				float3 isoSamplePos;
				isoSamplePos.x = (isoPos.x - boxMin.x) / diff.x;
				isoSamplePos.y = (isoPos.y - boxMin.y) / diff.y;
				isoSamplePos.z = (isoPos.z - boxMin.z) / diff.z;

				float3 gradient = make_float3(1, 0, 0);
				gradient.x = ((tex3D(tex, isoSamplePos.x + voxelSize.x, isoSamplePos.y, isoSamplePos.z) - minVal) / (maxVal - minVal))
					- ((tex3D(tex, isoSamplePos.x - voxelSize.x, isoSamplePos.y, isoSamplePos.z) - minVal) / (maxVal - minVal));
				gradient.y = ((tex3D(tex, isoSamplePos.x, isoSamplePos.y + voxelSize.y, isoSamplePos.z) - minVal) / (maxVal - minVal))
					- ((tex3D(tex, isoSamplePos.x, isoSamplePos.y - voxelSize.y, isoSamplePos.z) - minVal) / (maxVal - minVal));
				gradient.z = ((tex3D(tex, isoSamplePos.x, isoSamplePos.y, isoSamplePos.z + voxelSize.z) - minVal) / (maxVal - minVal))
					- ((tex3D(tex, isoSamplePos.x, isoSamplePos.y, isoSamplePos.z - voxelSize.z) - minVal) / (maxVal - minVal));
				gradient = normalize(gradient);

				float4 col = make_float4(0.0);

				// standard ray casting
				/*col = make_float4(sample);
				col.w *= density;*/

				// isosurface color
				col = colors[isoIndex];

				float4 mycol = performLighting(gradient, -eyeRay.d, lightDir, col, lightParams);
				// pre-multiply alpha
				mycol *= mycol.w;
				mycol.w = col.w;

				sum = sum + (mycol * (1.0f - sum.w));

				// exit early if opaque
				if (sum.w > opacityThreshold)
					break;

			}
			isoDiffsOld[isoIndex] = isoDiffs[isoIndex];
		}

		t += tstep;
		if (t > tfar) break;
		pos += step;
	}

	sum *= brightness;

	float depth = 0.0f;
	if (firstHit) {
		// there was no first hit, so we need the depth of the background
		depth = 1.0f;
	} else {
		// there was a first hit, compute the depth of the hit position
		float4 hpw = make_float4(firstHitPos, 1.0f);
		hpw = mul(c_mvpMatrix, hpw);
		float md = hpw.z / hpw.w;
		depth = 0.5 * md + 0.5;
	}

	// write output color
	d_output[y*imageW + x] = rgbaFloatToInt(sum);
	d_depth_output[y*imageW + x] = depth;
}

extern "C"
void setTextureFilterMode(bool bLinearFilter) {
	tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}

extern "C"
void initCudaDevice(void *h_volume, hipExtent volumeSize) {
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	// get min and max value from volume
	float *volptr = static_cast<float*>(h_volume);
	minVal = FLT_MAX;
	maxVal = FLT_MIN;
	for (unsigned int i = 0; i < volumeSize.width * volumeSize.height * volumeSize.depth; i++) {
		if (minVal > volptr[i])
			minVal = volptr[i];
		if (maxVal < volptr[i])
			maxVal = volptr[i];
	}
	//printf("min = %f, max = %f\n", minVal, maxVal);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	//std::cout << "Using 32-bit float volume data." << std::endl;
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeMirror; // we want 0s outside for beautiful isosurfaces at the borders
	tex.addressMode[1] = hipAddressModeMirror;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

	float4 customTransFunc[256];
	for (int i = 0; i < 256; i++)
		customTransFunc[i] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
	hipArray *d_customTransferFuncArray;
	checkCudaErrors(hipMallocArray(&d_customTransferFuncArray, &channelDesc3, 256, 1));
	checkCudaErrors(hipMemcpyToArray(d_customTransferFuncArray, 0, 0, customTransFunc, sizeof(customTransFunc), hipMemcpyHostToDevice));

	customTransferTex.filterMode = hipFilterModeLinear;
	customTransferTex.normalized = true;
	customTransferTex.addressMode[0] = hipAddressModeClamp;

	checkCudaErrors(hipBindTextureToArray(customTransferTex, d_customTransferFuncArray, channelDesc3));

	checkCudaErrors(hipEventCreate(&evtStart));
	checkCudaErrors(hipEventCreate(&evtStop));
}

extern "C"
void freeCudaBuffers() {
	checkCudaErrors(hipFreeArray(d_volumeArray));
	checkCudaErrors(hipFreeArray(d_customTransferFuncArray));
	//checkCudaErrors(hipFreeArray(d_isoValArray));
	//checkCudaErrors(hipFree(d_isoValArray));
}


extern "C"
void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output, float *d_depth_output, uint imageW, uint imageH, float fovx, float fovy, float3 camPos, float3 camDir,
	float3 camUp, float3 camRight, float zNear, float density, float brightness, float transferOffset, float transferScale,
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f), const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f), hipExtent volSize = make_hipExtent(1, 1, 1),
	const float3 lightDir = make_float3(1.0f, 1.0f, 1.0f), const float4 lightParams = make_float4(0.3f, 0.5f, 0.4f, 10.0f),
	const float4 plane = make_float4(0.0f, 0.0f, 0.0f, 0.0), bool enablePlane = false) {

	d_render<<<gridSize, blockSize>>>(d_output, d_depth_output, imageW, imageH, fovx, fovy, camPos, camDir, camUp, camRight, zNear, density,
		brightness, transferOffset, transferScale, minVal, maxVal, boxMin, boxMax, volSize, lightDir, lightParams, plane, enablePlane);
}

extern "C"
void renderArray_kernel(hipArray* renderArray, dim3 gridSize, dim3 blockSize, uint *d_output, float * d_depth_output, uint imageW, uint imageH, float fovx, float fovy, float3 camPos, float3 camDir,
	float3 camUp, float3 camRight, float zNear, float density, float brightness, float transferOffset, float transferScale,
	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f), const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f), hipExtent volSize = make_hipExtent(1,1,1),
	const float3 lightDir = make_float3(1.0f, 1.0f, 1.0f), const float4 lightParams = make_float4(0.3f, 0.5f, 0.4f, 10.0f),
	const float4 plane = make_float4(0.0f, 0.0f, 0.0f, 0.0), bool enablePlane = false) {
	
	float* horst = new float[volSize.width * volSize.height * volSize.depth];
	horst[0] = 1.0f;
	checkCudaErrors(hipDeviceSynchronize());
	//checkCudaErrors(hipMemcpyFromArray(horst, renderArray, 0, 0, sizeof(float) * volSize.x * volSize.y, hipMemcpyDeviceToHost));
	
	hipExtent volExt = volSize;

	hipPitchedPtr pitchedHorst = make_hipPitchedPtr(horst, sizeof(float) * volSize.width, volSize.height, volSize.height);

	hipMemcpy3DParms myParms = { 0 };
	myParms.extent = volExt;
	myParms.srcArray = renderArray;
	myParms.dstPtr = pitchedHorst;
	myParms.kind = hipMemcpyDeviceToHost;

	checkCudaErrors(hipMemcpy3D(&myParms));

	checkCudaErrors(hipDeviceSynchronize());
	for (unsigned int i = 0; i < volSize.width * volSize.depth * volSize.height; i++) {
		if (horst[i] > 0.000001)
			printf("%i - %.3f\n", i, horst[i]);
	}
	delete[] horst;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipBindTextureToArray(tex, renderArray, channelDesc));

	checkCudaErrors(hipDeviceSynchronize());

	d_render << <gridSize, blockSize >> >(d_output, d_depth_output, imageW, imageH, fovx, fovy, camPos, camDir, camUp, camRight, zNear, density,
		brightness, transferOffset, transferScale, minVal, maxVal, boxMin, boxMax, volSize, lightDir, lightParams, plane, enablePlane);

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipUnbindTexture(tex));
}

extern "C"
void copyLUT(float4* myLUT, int lutSize = 256)
{
	checkCudaErrors(hipFreeArray(d_customTransferFuncArray));

	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float4>();
	hipArray *d_customTransferFuncArray;
	checkCudaErrors(hipMallocArray(&d_customTransferFuncArray, &channelDesc3, 256, 1));
	checkCudaErrors(hipMemcpyToArray(d_customTransferFuncArray, 0, 0, myLUT, sizeof(float4)* lutSize, hipMemcpyHostToDevice));

	customTransferTex.filterMode = hipFilterModeLinear;
	customTransferTex.normalized = true;
	customTransferTex.addressMode[0] = hipAddressModeClamp;

	checkCudaErrors(hipBindTextureToArray(customTransferTex, d_customTransferFuncArray, channelDesc3));
}

extern "C"
void transferIsoValues(float4 h_isoVals, int h_numIsos) {

	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_isoVals), &h_isoVals, sizeof(float4), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_iso1), &(h_isoVals.x), sizeof(float), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_iso2), &(h_isoVals.y), sizeof(float), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_iso3), &(h_isoVals.z), sizeof(float), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_iso4), &(h_isoVals.w), sizeof(float), 0, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_numIsoVals), &h_numIsos, sizeof(int), 0, hipMemcpyHostToDevice));
}

extern "C" 
void copyMVPMatrix(float * mvp, size_t sizeofMatrix) {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_mvpMatrix), mvp, sizeofMatrix));
	checkCudaErrors(hipDeviceSynchronize());
}

extern "C"
void transferNewVolume(void* h_volume, hipExtent volumeSize) {

	if (d_volumeArray) {
		checkCudaErrors(hipFreeArray(d_volumeArray));
		d_volumeArray = 0;

		checkCudaErrors(hipUnbindTexture(tex));
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	// get min and max value from volume
	float *volptr = static_cast<float*>(h_volume);
	thrust::device_ptr<float> ptr = thrust::device_ptr<float>(volptr);

	auto res = thrust::minmax_element(ptr, ptr + (volumeSize.width * volumeSize.depth * volumeSize.height));
	minVal = (float)*res.first;
	maxVal = (float)*res.second;

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);

	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeMirror;
	tex.addressMode[1] = hipAddressModeMirror;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

extern "C"
void transferVolumeDirect(void * h_volume, hipExtent volumeSize, float myMin, float myMax) {

	if (d_volumeArray) {
		checkCudaErrors(hipFreeArray(d_volumeArray));
		d_volumeArray = 0;

		checkCudaErrors(hipUnbindTexture(tex));
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

	//checkCudaErrors(hipDeviceSynchronize());

	minVal = myMin;
	maxVal = myMax;

	//printf("min %f; max %f\n", minVal, maxVal);
	//printf("%Iu %Iu %Iu\n", volumeSize.width, volumeSize.height, volumeSize.depth);

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width*sizeof(VolumeType), volumeSize.width, volumeSize.height);

	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeMirror;
	tex.addressMode[1] = hipAddressModeMirror;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
