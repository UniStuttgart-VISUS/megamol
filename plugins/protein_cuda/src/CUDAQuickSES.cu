#include "CUDAQuickSES.h"


CUDAQuickSES::CUDAQuickSES() {
	// Query GPU device attributes so we can launch the best kernel type
	hipDeviceProp_t deviceProp;
	memset(&deviceProp, 0, sizeof(hipDeviceProp_t));

	if (hipGetDevice(&cudadevice) != hipSuccess) {
		// XXX do something more useful here...
	}

	if (hipGetDeviceProperties(&deviceProp, cudadevice) != hipSuccess) {
		// XXX do something more useful here...
	}

	cudacomputemajor = deviceProp.major;
}


CUDAQuickSES::~CUDAQuickSES() {

}
