#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* 
 * CUDA particle system kernel code.
 */

#ifndef _PARTICLES_KERNEL_H_
#define _PARTICLES_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include "hip/hip_math_constants.h"
#include "particles_kernel.cuh"
#include "hip/hip_vector_types.h"

// texture for particle position
texture<float4, 1, hipReadModeElementType> oldPosTex;
// texture for atom position
texture<float4, 1, hipReadModeElementType> atomPosTex;
// texture for number of neighbor atoms
texture<uint, 1, hipReadModeElementType> neighborCountTex;
// texture for neighbor atoms (indices)
texture<uint, 1, hipReadModeElementType> neighborsTex;
// texture for small circles (vector to center 
texture<float4, 1, hipReadModeElementType> smallCirclesTex;
// texture for arcs
texture<float4, 1, hipReadModeElementType> arcsTex;

texture<uint, 1, hipReadModeElementType> gridParticleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<uint, 1, hipReadModeElementType> cellEndTex;

texture<float4, 1, hipReadModeElementType> visibleAtomsTex;
texture<uint, 1, hipReadModeElementType> visibleAtomsIdTex;
//texture<uint4, 1, hipReadModeElementType> point1Tex;

texture<float4, 2, hipReadModeElementType> inVisibilityTex;

// simulation parameters in constant memory
__constant__ SimParams params;

// Reduced Surfaec parameters in constant memory
__constant__ RSParams rsParams;

///////////////////////////////////////////////////////////////////////////////
// calculate position in uniform grid
///////////////////////////////////////////////////////////////////////////////
__device__ int3 calcGridPos(float3 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.worldOrigin.x) / params.cellSize.x);
    gridPos.y = floor((p.y - params.worldOrigin.y) / params.cellSize.y);
    gridPos.z = floor((p.z - params.worldOrigin.z) / params.cellSize.z);
    return gridPos;
}

///////////////////////////////////////////////////////////////////////////////
// calculate address in grid from position (clamping to edges)
///////////////////////////////////////////////////////////////////////////////
__device__ uint calcGridHash(int3 gridPos)
{
    gridPos.x = gridPos.x & (params.gridSize.x-1);  // wrap grid, assumes size is power of 2
    gridPos.y = gridPos.y & (params.gridSize.y-1);
    gridPos.z = gridPos.z & (params.gridSize.z-1);
    return __umul24(__umul24(gridPos.z, params.gridSize.y), params.gridSize.x) + __umul24(gridPos.y, params.gridSize.x) + gridPos.x;
}

///////////////////////////////////////////////////////////////////////////////
// calculate grid hash value for each particle
///////////////////////////////////////////////////////////////////////////////
__global__
void calcHashD(uint*   gridParticleHash,  // output
               uint*   gridParticleIndex, // output
               float4* pos,               // input: positions
               uint    numParticles)
{
    uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;
    
    volatile float4 p = pos[index];

    // get address in grid
    int3 gridPos = calcGridPos(make_float3(p.x, p.y, p.z));
    uint hash = calcGridHash(gridPos);

    // store grid hash and particle index
    gridParticleHash[index] = hash;
    gridParticleIndex[index] = index;
}

///////////////////////////////////////////////////////////////////////////////
// rearrange particle data into sorted order, and find the start of each cell
// in the sorted hash array
///////////////////////////////////////////////////////////////////////////////
__global__
void reorderDataAndFindCellStartD(uint*   cellStart,        // output: cell start index
                                  uint*   cellEnd,          // output: cell end index
                                  float4* sortedPos,        // output: sorted positions
                                  uint *  gridParticleHash, // input: sorted grid hashes
                                  uint *  gridParticleIndex,// input: sorted particle indices
                                  float4* oldPos,           // input: sorted position array
                                  uint    numParticles)
{
    extern __shared__ uint sharedHash[];    // blockSize + 1 elements
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    
    uint hash;
    // handle case when no. of particles not multiple of block size
    if (index < numParticles) {
        hash = gridParticleHash[index];

        // Load hash data into shared memory so that we can look 
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if (index > 0 && threadIdx.x == 0)
        {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridParticleHash[index-1];
        }
    }

    __syncthreads();
    
    if( index < numParticles ) {
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell

        if (index == 0 || hash != sharedHash[threadIdx.x]) {
            cellStart[hash] = index;
            if (index > 0)
                cellEnd[sharedHash[threadIdx.x]] = index;
        }

        if (index == numParticles - 1) {
            cellEnd[hash] = index + 1;
        }

        // Now use the sorted index to reorder the pos data
        uint sortedIndex = gridParticleIndex[index];
        //float4 pos = FETCH( oldPos, sortedIndex);       // macro does either global read or texture fetch
        float4 pos = oldPos[sortedIndex];       // Do not use texture for large arrays!

        sortedPos[index] = pos;
    }
}

///////////////////////////////////////////////////////////////////////////////
// count all neighbor atoms in a given cell
///////////////////////////////////////////////////////////////////////////////
__device__
uint countNeighborsInCell( uint*   neighbors,     // output: neighbor indices
                           float4* smallCircles,  // output: small circles
                           uint    neighborIndex, // input: first index for writing in neighbor list
                           uint    atomIndex,     // input: atom index for writing in neighbor list
                           int3    gridPos,
                           uint    index,
                           float4  pos,
                           float4* atomPos,
                           uint*   gridParticleIndex,    // input: sorted atom indices
                           uint*   cellStart,
                           uint*   cellEnd) {
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH( cellStart, gridHash);

    uint count = 0;
    float4 pos2;
    float3 relPos;
    float dist;
    float neighborDist;
    float r;
    float3 vec;
    float4 smallCircle;
    if( startIndex != 0xffffffff ) {	// cell is not empty
        // iterate over atoms in this cell
        uint endIndex = FETCH( cellEnd, gridHash);
        for( uint j = startIndex; j < endIndex; j++) {
            // do not count self
            if( j != index) {
                // get position of potential neighbor
                pos2 = FETCH( atomPos, j);
                // check distance
                relPos = make_float3( pos2.x, pos2.y, pos2.z) - make_float3( pos.x, pos.y, pos.z);
                dist = length( relPos);
                neighborDist = pos.w + pos2.w + 2.0f * params.probeRadius;
                if( dist < neighborDist ) {
                    // check number of neighbors
                    if( ( neighborIndex + count) >= params.maxNumNeighbors ) return count;
                    //neighbors[atomIndex*params.maxNumNeighbors+neighborIndex+count] = gridParticleIndex[j];
                    neighbors[atomIndex*params.maxNumNeighbors+neighborIndex+count] = j;
                    // compute small circle / intersection plane
                    r = ( (pos.w + params.probeRadius)*(pos.w + params.probeRadius))
                        + ( dist * dist) //dot( relPos, relPos) 
                        - ( (pos2.w + params.probeRadius)*(pos2.w + params.probeRadius));
                    r = r / (2.0f * dist * dist); // dot( relPos, relPos));
                    /*
                    r = (pos.w + params.probeRadius)*(pos.w + params.probeRadius)
                        - (pos2.w + params.probeRadius)*(pos2.w + params.probeRadius);
                    r = r / (2.0f * dot( relPos, relPos));
                    r = r + 0.5f;
                    */
                    vec = relPos * r;
                    smallCircle.x = vec.x;
                    smallCircle.y = vec.y;
                    smallCircle.z = vec.z;
                    smallCircle.w = 1.0f;
                    smallCircles[atomIndex*params.maxNumNeighbors+neighborIndex+count] = smallCircle;
                    // increment the neighbor counter
                    count++;
                }
            }
        }
    }
    return count;
}

///////////////////////////////////////////////////////////////////////////////
// count all neighbor atoms in a given cell
///////////////////////////////////////////////////////////////////////////////
__device__
uint countNeighborsInCell( uint*   neighbors,     // output: neighbor indices
                           uint    neighborIndex, // input: first index for writing in neighbor list
                           uint    atomIndex,     // input: atom index for writing in neighbor list
                           int3    gridPos,
                           uint    index,
                           float4  pos,
                           float4* atomPos,
                           uint*   gridParticleIndex,    // input: sorted atom indices
                           uint*   cellStart,
                           uint*   cellEnd) {
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH( cellStart, gridHash);

    uint count = 0;
    float4 pos2;
    float3 relPos;
    float dist;
    float neighborDist;
    if( startIndex != 0xffffffff ) {	// cell is not empty
        // iterate over atoms in this cell
        uint endIndex = FETCH( cellEnd, gridHash);
        for( uint j = startIndex; j < endIndex; j++) {
            // do not count self
            if( j != index) {
                // get position of potential neighbor
                pos2 = FETCH( atomPos, j);
                // check distance
                relPos = make_float3( pos2.x,  pos2.y,  pos2.z) - make_float3( pos.x, pos.y, pos.z);
                dist = length( relPos);
                neighborDist = pos.w + pos2.w + 2.0f * params.probeRadius;
                if( dist < neighborDist ) {
                    // check number of neighbors
                    if( ( neighborIndex + count) >= params.maxNumNeighbors ) return count;
                    //neighbors[atomIndex*params.maxNumNeighbors+neighborIndex+count] = gridParticleIndex[j];
                    neighbors[atomIndex*params.maxNumNeighbors+neighborIndex+count] = j;
                    // increment the neighbor counter
                    count++;
                }
            }
        }
    }
    return count;
}

///////////////////////////////////////////////////////////////////////////////
// count all neighbor atoms in a given cell
///////////////////////////////////////////////////////////////////////////////
__device__
uint countProbeNeighborsInCell( //uint*   neighbors,     // output: neighbor indices
                           float3* neighbors,     // output: neighbor positions
                           uint    neighborIndex, // input: first index for writing in neighbor list
                           uint    atomIndex,     // input: atom index for writing in neighbor list
                           int3    gridPos,
                           uint    index,
                           float4  pos,
                           float4* atomPos,
                           uint*   gridParticleIndex,    // input: sorted atom indices
                           uint*   cellStart,
                           uint*   cellEnd) {
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH( cellStart, gridHash);

    uint count = 0;
    float4 pos2;
    float3 relPos;
    float dist;
    float neighborDist;
    if( startIndex != 0xffffffff ) {	// cell is not empty
        // iterate over atoms in this cell
        uint endIndex = FETCH( cellEnd, gridHash);
        for( uint j = startIndex; j < endIndex; j++) {
            // do not count self
            if( j != index) {
                // get position of potential neighbor
                pos2 = atomPos[j];
                // check distance
                relPos = make_float3( pos2.x, pos2.y, pos2.z) - make_float3( pos.x, pos.y, pos.z);
                dist = length( relPos);
                neighborDist = 2.0f * params.probeRadius;
                if( dist < neighborDist ) {
                    // check number of neighbors
                    if( ( neighborIndex + count) >= rsParams.maxNumProbeNeighbors ) return count;
                    neighbors[atomIndex*rsParams.maxNumProbeNeighbors+neighborIndex+count] = make_float3( pos2.x, pos2.y, pos2.z);
                    // increment the neighbor counter
                    count++;
                }
            }
        }
    }
    return count;
}

///////////////////////////////////////////////////////////////////////////////
// counting function
///////////////////////////////////////////////////////////////////////////////
__global__
void countNeighbors( uint*   neighborCount,        // output: number of neighbors
                     uint*   neighbors,            // output: neighbor indices
                     float4* smallCircles,         // output: small circles
                     float4* atomPos,              // input: sorted atom positions
                     uint*   gridParticleIndex,    // input: sorted atom indices
                     uint*   cellStart,
                     uint*   cellEnd,
                     uint    numAtoms) {
    uint index = __mul24( blockIdx.x, blockDim.x) + threadIdx.x;
    if( index >= numAtoms ) return;
    
    // read original unsorted atom location
    uint originalIndex = gridParticleIndex[index];

    // read atom data from sorted arrays
    float4 pos = FETCH( atomPos, index);

    // get address in grid
    int3 gridPos = calcGridPos( make_float3( pos.x, pos.y, pos.z));

    int3 gridSize;
    gridSize.x = int( params.gridSize.x);
    gridSize.y = int( params.gridSize.y);
    gridSize.z = int( params.gridSize.z);
    // search range for neighbor atoms: max atom diameter + probe diameter
    float range = ( pos.w + 3.0f + 2.0f * params.probeRadius);
    // compute number of grid cells
    int3 cellsInRange;
    cellsInRange.x = ceil( range / params.cellSize.x);
    cellsInRange.y = ceil( range / params.cellSize.y);
    cellsInRange.z = ceil( range / params.cellSize.z);
    int3 start = gridPos - cellsInRange;
    int3 end = gridPos + cellsInRange;

    // examine neighbouring cells
    uint count = 0;
    int3 neighborPos;
    for( int z = ( start.z > 0 ? start.z : 0); z < ( end.z > gridSize.z ? gridSize.z : end.z) ; z++ ) {
        for( int y = ( start.y > 0 ? start.y : 0); y < ( end.y > gridSize.y ? gridSize.y : end.y) ; y++ ) {
            for( int x = ( start.x > 0 ? start.x : 0); x < ( end.x > gridSize.x ? gridSize.x : end.x) ; x++ ) {
                neighborPos = make_int3( x, y, z);
                count += countNeighborsInCell( neighbors, smallCircles, count, originalIndex, neighborPos, index, pos, atomPos, gridParticleIndex, cellStart, cellEnd);
            }
        }
    }

    // write new neighbor atom count back to original unsorted location
    neighborCount[originalIndex] = count;
}

///////////////////////////////////////////////////////////////////////////////
// counting function
///////////////////////////////////////////////////////////////////////////////
__global__
void countNeighbors( uint*   neighborCount,        // output: number of neighbors
                     uint*   neighbors,            // output: neighbor indices
                     float4* atomPos,              // input: sorted atom positions
                     uint*   gridParticleIndex,    // input: sorted atom indices
                     uint*   cellStart,
                     uint*   cellEnd,
                     uint    numAtoms) {
    uint index = __mul24( blockIdx.x, blockDim.x) + threadIdx.x;
    if( index >= numAtoms ) return;
    
    // read original unsorted atom location
    uint originalIndex = gridParticleIndex[index];

    // read atom data from sorted arrays
    float4 pos = FETCH( atomPos, index);

    // get address in grid
    int3 gridPos = calcGridPos( make_float3( pos));

    int3 gridSize;
    gridSize.x = int( params.gridSize.x);
    gridSize.y = int( params.gridSize.y);
    gridSize.z = int( params.gridSize.z);
    // search range for neighbor atoms: max atom diameter + probe diameter
    float range = ( pos.w + 3.0f + 2.0f * params.probeRadius);
    // compute number of grid cells
    int3 cellsInRange;
    cellsInRange.x = ceil( range / params.cellSize.x);
    cellsInRange.y = ceil( range / params.cellSize.y);
    cellsInRange.z = ceil( range / params.cellSize.z);
    int3 start = gridPos - cellsInRange;
    int3 end = gridPos + cellsInRange;

    // examine neighbouring cells
    uint count = 0;
    int3 neighborPos;
    for( int z = ( start.z > 0 ? start.z : 0); z < ( end.z > gridSize.z ? gridSize.z : end.z) ; z++ ) {
        for( int y = ( start.y > 0 ? start.y : 0); y < ( end.y > gridSize.y ? gridSize.y : end.y) ; y++ ) {
            for( int x = ( start.x > 0 ? start.x : 0); x < ( end.x > gridSize.x ? gridSize.x : end.x) ; x++ ) {
                neighborPos = make_int3( x, y, z);
                count += countNeighborsInCell( neighbors, count, originalIndex, neighborPos, index, pos, atomPos, gridParticleIndex, cellStart, cellEnd);
            }
        }
    }

    // write new neighbor atom count back to original unsorted location
    neighborCount[originalIndex] = count;
}

///////////////////////////////////////////////////////////////////////////////
// counting function
///////////////////////////////////////////////////////////////////////////////
__global__
void countProbeNeighbors( //uint*   probeNeighborCount, // output: number of neighbors
                     float3* probeNeighborCount,        // output: number of neighbors
                     //uint*   probeNeighbors,          // output: neighbor indices
                     float3* probeNeighbors,            // output: neighbor indices
                     float4* probePos,                  // input: sorted atom positions
                     uint*   gridParticleIndex,         // input: sorted atom indices
                     uint*   cellStart,
                     uint*   cellEnd,
                     uint    numProbes) {
    uint index = __mul24( blockIdx.x, blockDim.x) + threadIdx.x;
    if( index >= numProbes ) return;
    
    // read original unsorted atom location
    uint originalIndex = gridParticleIndex[index];

    // read atom data from sorted arrays
    float4 pos = probePos[index];

    // get address in grid
    int3 gridPos = calcGridPos( make_float3( pos.x, pos.y, pos.z));

    int3 gridSize;
    gridSize.x = int( params.gridSize.x);
    gridSize.y = int( params.gridSize.y);
    gridSize.z = int( params.gridSize.z);
    // search range for neighbor probes: 2x probe diameter
    float range = 2.0f * params.probeRadius;
    // compute number of grid cells
    int3 cellsInRange;
    cellsInRange.x = ceil( range / params.cellSize.x);
    cellsInRange.y = ceil( range / params.cellSize.y);
    cellsInRange.z = ceil( range / params.cellSize.z);
    int3 start = gridPos - cellsInRange;
    int3 end = gridPos + cellsInRange;

    // examine neighbouring cells
    uint count = 0;
    int3 neighborPos;
    for( int z = ( start.z > 0 ? start.z : 0); z < ( end.z > gridSize.z ? gridSize.z : end.z) ; z++ ) {
        for( int y = ( start.y > 0 ? start.y : 0); y < ( end.y > gridSize.y ? gridSize.y : end.y) ; y++ ) {
            for( int x = ( start.x > 0 ? start.x : 0); x < ( end.x > gridSize.x ? gridSize.x : end.x) ; x++ ) {
                neighborPos = make_int3( x, y, z);
                count += countProbeNeighborsInCell( probeNeighbors, count, originalIndex, neighborPos, index, pos, probePos, gridParticleIndex, cellStart, cellEnd);
            }
        }
    }

    // write new neighbor atom count back to original unsorted location
    probeNeighborCount[originalIndex] = make_float3( float( count), 0.0f, float( originalIndex));
}

///////////////////////////////////////////////////////////////////////////////
// compute the arcs
///////////////////////////////////////////////////////////////////////////////
__global__
void computeArcs( float4* arcs,                 // output: arcs
                  uint*   neighborCount,        // input: number of neighbors
                  uint*   neighbors,            // input: neighbor indices
                  float4* smallCircles,         // input: small circles
                  float4* atomPos,              // input: sorted atom positions
                  uint*   gridParticleIndex,    // input: sorted atom indices
                  uint    numAtoms) {
    // get atom index
    uint atomIdx = blockIdx.x;
    // get neighbor atom index
    uint neighborIdx = threadIdx.x;
    // check, if atom index is within bounds
    if( atomIdx >= numAtoms ) return;
    // check, if neighbor index is within bounds
    if( neighborIdx >= params.maxNumNeighbors ) return;
    // read original unsorted atom location
    uint origAtomIdx = gridParticleIndex[atomIdx];
    // check, if neighbor index is within bounds
    uint numNeighbors = neighborCount[origAtomIdx];
    if( neighborIdx >= numNeighbors ) return;

    // read atom position from sorted arrays
    float4 atom = FETCH( atomPos, atomIdx);
    // read neighbor position from sorted arrays
    //float4 ak = FETCH( atomPos, FETCH( neighbors, origAtomIdx * params.maxNumNeighbors + neighborIdx));
    float3 ai = make_float3( atom);
    float3 aj;

    float3 rm;
    float4 rj4;
    float3 rj;
    float4 rk4 = FETCH( smallCircles, origAtomIdx * params.maxNumNeighbors + neighborIdx);
    float3 rk = make_float3( rk4.x, rk4.y, rk4.z);
    float Ri = atom.w + params.probeRadius;
    float Ri2 = Ri * Ri;
    float numer1, numer2, denom, rj_dot_rk, rj2, rk2;
    float3 p1, p2, tmpFloat3, cross_rj_rk;
    uint numArcs = 0;

    rk2 = dot( rk, rk);

    float3 e1, e2;
    
    for( uint cnt = 0; cnt < numNeighbors; ++cnt ) {
        if( cnt == neighborIdx ) continue;
        
        aj = make_float3( FETCH( atomPos, FETCH( neighbors, origAtomIdx * params.maxNumNeighbors + cnt)));
        // compute the auxiliary vector rm (plane intersection)
        rj4 = FETCH( smallCircles, origAtomIdx * params.maxNumNeighbors + cnt);
        rj = make_float3( rj4);
        rj2 = dot( rj, rj);
        rj_dot_rk = dot( rj, rk);
        numer1 = ( rj2 - rj_dot_rk) * rk2;
        numer2 = ( rk2 - rj_dot_rk) * rj2;
        denom = rj2 * rk2 - rj_dot_rk * rj_dot_rk;
        rm = rj * ( numer1 / denom) + rk * ( numer2 / denom);

        // continue to next small circle, if this one does not intersect
        if( dot( rm, rm) > Ri2 ) continue;

        // compute the start- and endpoint of the newly found arc
        cross_rj_rk = cross( rj, rk);
        tmpFloat3 = cross_rj_rk * sqrt( ( Ri2 - dot( rm, rm)) / dot( cross_rj_rk, cross_rj_rk));
        
        if( dot( rj, aj - ai) < 0.0f ) {
            // x1
            p1 = rm + tmpFloat3;
            // x2
            p2 = rm - tmpFloat3;
        } else {
            // x2
            p2 = rm + tmpFloat3;
            // x1
            p1 = rm - tmpFloat3;
        }

        if( dot( cross( p1 - rk, p2 - rk), rk) < 0.0f ) {
            tmpFloat3 = p1;
            p1 = p2;
            p2 = tmpFloat3;
        }
        
        // if the current arc ist the first:
        if( numArcs == 0 ) {
            // write the first arc
            //arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p1, 1.0f);
            //arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
            numArcs++;

            e1 = p1;
            e2 = p2;
            continue;
        }

        // compute angles
        /*
        tmpAngle1 = acos( dot( normalize( p1 - rk), e1));
        tmpAngle = dot( normalize( p1 - rk), e2);
        if( tmpAngle < 0.0f )
            tmpAngle1 = 6.2831853f - tmpAngle1;
        tmpAngle2 = acos( dot( normalize( p2 - rk), e1));
        tmpAngle = dot( normalize( p2 - rk), e2);
        if( tmpAngle < 0.0f )
            tmpAngle2 = 6.2831853f - tmpAngle2;

        // check cases
        if( tmpAngle1 > angle1 ) {
            angle1 = tmpAngle1;
            arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p1, 1.0f);
        }
        if( tmpAngle2 < angle2 ) {
            angle2 = tmpAngle2;
            arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
        }
        if( tmpAngle1 < angle2 && tmpAngle2 > angle1 ) {
            angle2 = tmpAngle2;
            arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
            // TODO: second arc segment
        }
        */
        float d1 = dot( aj - ai, e1 - rj);
        float d2 = dot( aj - ai, e2 - rj);
        float d3 = dot( rk, aj - ai) * dot( cross( e1, p1), e2);

        if( d1 > 0.0f ) {
            if( d2 > 0.0f ) {
                if( d3 > 0.0f ) {
                    e1 = make_float3( 0.0f, 0.0f, 0.0f);
                    e2 = make_float3( 0.0f, 0.0f, 0.0f);
                } else { // d3 < 0
                    e1 = p1;
                    e2 = p2;
                }
            } else { // d2 < 0
                if( d3 > 0.0f ) {
                    e1 = p1;
                } else { // d3 < 0
                    e1 = p2;
                }
            }
        } else { // d1 < 0
            if( d2 > 0.0f ) {
                if( d3 > 0.0f ) {
                    e2 = p1;
                } else { // d3 < 0
                    e2 = p2;
                }
            } else { // d2 < 0
                if( d3 > 0.0f ) {
                    // Teilung in zwei B�gen
                } else { // d3 < 0
                    // keine Auswirkung
                }
            }
        }
    }
    arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( e1, 1.0f);
    arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( e2, 1.0f);

    /*
    for( uint cnt = 0; cnt < numNeighbors; ++cnt ) {
        if( cnt == neighborIdx ) continue;
        
        //aj = FETCH( atomPos, FETCH( neighbors, origAtomIdx * params.maxNumNeighbors + cnt));
        // do nothing if the neighboring atoms do not intersect
        //if( length( make_float3( aj) - make_float3( ak)) > ( ak.w + aj.w + 2.0f * params.probeRadius) ) continue;

        // compute the auxiliary vector rm (plane intersection)
        rj4 = FETCH( smallCircles, origAtomIdx * params.maxNumNeighbors + cnt);
        rj = make_float3( rj4);
        rj2 = dot( rj, rj);
        rj_dot_rk = dot( rj, rk);
        numer1 = ( rj2 - rj_dot_rk) * rk2;
        numer2 = ( rk2 - rj_dot_rk) * rj2;
        denom = rj2 * rk2 - rj_dot_rk * rj_dot_rk;
        rm = rj * ( numer1 / denom) + rk * ( numer2 / denom);

        // continue to next small circle, if this one does not intersect
        if( dot( rm, rm) > Ri2 ) continue;

        // compute the start- and endpoint of the newly found arc
        cross_rj_rk = cross( rj, rk);
        tmpFloat3 = cross_rj_rk * sqrt( ( Ri2 - dot( rm, rm)) / dot( cross_rj_rk, cross_rj_rk));
        p1 = rm + tmpFloat3;
        p2 = rm - tmpFloat3;

        // COMPUTATION WITH GLOBAL MEMORY ...
        // if the current arc ist the first:
        if( numArcs == 0 ) {
            // write the first arc
            arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p1, 1.0f);
            numArcs++;
            arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
            numArcs++;
            sk = 1.0f;
            continue;
        }

        // compute c1, c2, d to determine the case
        c1 = dot( rj, make_float3( arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0]) - rj);
        c2 = dot( rj, make_float3( arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1]) - rj);
        d = sk * dot( cross( make_float3( arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0]), p1), make_float3( arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1]));

        if( c1 > 0.0f && c2 < 0.0f ) {
            if( d > 0.0f ) {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p1, 1.0f);
            } else {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p2, 1.0f);
            }
        } else if( c1 < 0.0f && c2 > 0.0f ) {
            if( d > 0.0f ) {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p1, 1.0f);
            } else {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
            }
        } else if( c1 > 0.0f && c2 > 0.0f ) {
            if( d > 0.0f ) {
                return;
            } else {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 0] = make_float4( p1, 1.0f);
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p2, 1.0f);
            }
        }
        else { // c1 < 0.0f && c2 < 0.0f
            if( d > 0.0f ) {
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 2] = arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1];
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 1] = make_float4( p1, 1.0f);
                arcs[origAtomIdx * params.maxNumNeighbors * 4 + 3] = make_float4( p2, 1.0f);
            }
        }
        // ... COMPUTATION WITH GLOBAL MEMORY
    }
    */

}


///////////////////////////////////////////////////////////////////////////////
// compute reduced surfaces
///////////////////////////////////////////////////////////////////////////////
__global__
void computeReducedSurface( uint4* point1,      // output: the atom indices
                  float4* probePos,             // output: the probe position and orientation
                  uint*   neighborCount,        // input: number of neighbors
                  uint*   neighbors,            // input: neighbor indices
                  float4* atomPos,              // input: sorted atom positions
                  uint*   gridParticleIndex,    // input: sorted atom indices
                  float4* visibleAtoms,         // input: visible atoms position and radius
                  uint*   visibleAtomsId ) {    // input: visible atoms original index list

    // get atom index
    uint visibleAtomIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // check bounds of visible atoms
    if( visibleAtomIdx >= rsParams.visibleAtomCount ) {
        return;
    }
    // get combined neighbor index
    uint idxX = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds of neighbor index
    if( idxX >= ( params.maxNumNeighbors * params.maxNumNeighbors) ) {
        return;
    }
    // get neighbor atom indices
    uint id1 = idxX % params.maxNumNeighbors;
    uint id2 = ( idxX - id1) / params.maxNumNeighbors;
    // check if the id's are the same
    if( id1 == id2 ) {
        return;
    }
    
    // ---------- FRAGMENT SHADER CODE ... ---------
    
    // get the Id of the current atom
    float atomId = FETCH( visibleAtomsId, visibleAtomIdx);

    // read number of vicinity atoms from vicinity texture
    uint vicinityCnt = FETCH( neighborCount, atomId);

    // temp variables
    int cutId;

    float4 akTmp = FETCH( visibleAtoms, visibleAtomIdx);
    uint aiIdx = FETCH( neighbors, params.maxNumNeighbors * atomId + id1);
    float4 aiTmp = FETCH( atomPos, aiIdx);
    uint ajIdx = FETCH( neighbors, params.maxNumNeighbors * atomId + id2);
    float4 ajTmp = FETCH( atomPos, ajIdx);
    float3 ak = { akTmp.x, akTmp.y, akTmp.z}; 
    float rk = akTmp.w;
    float3 ai = { aiTmp.x, aiTmp.y, aiTmp.z};
    float ri = aiTmp.w;
    float3 aj = { ajTmp.x, ajTmp.y, ajTmp.z};
    float rj = ajTmp.w;
    
    // names of the variables according to: Connolly "Analytical Molecular Surface Calculation", 1983
    float3 uij, uik, tij, tik, uijk, utb, bijk, pijk0, pijk1;
    float dij, dik, djk, hijk, wijk, tmpFloat;
    
    dij = length( aj - ai);
    dik = length( ak - ai);
    djk = length( ak - aj);
    
    uij = ( aj - ai)/dij;
    uik = ( ak - ai)/dik;

    if( ( ( ri - rj)*( ri - rj) > dij*dij ) || 
        ( ( ri - rk)*( ri - rk) > dik*dik ) || 
        ( ( rj - rk)*( rj - rk) > djk*djk ) ) {
        return;
    }
    tij = 0.5f*( ai + aj) + 0.5f*( aj - ai) * ( ( ri + params.probeRadius)*( ri + params.probeRadius) - ( rj + params.probeRadius)*( rj + params.probeRadius))/( dij*dij);
    tik = 0.5f*( ai + ak) + 0.5f*( ak - ai) * ( ( ri + params.probeRadius)*( ri + params.probeRadius) - ( rk + params.probeRadius)*( rk + params.probeRadius))/( dik*dik);
    wijk = acos( dot( uij, uik) );
    uijk = cross( uij, uik) / sin( wijk);
    utb = cross( uijk, uij);
    bijk = tij + utb * ( dot( uik, tik - tij) / sin( wijk));
    tmpFloat = ( ri + params.probeRadius)*( ri + params.probeRadius) - length( bijk - ai)*length( bijk - ai);
    if( tmpFloat < 0.0f ) {
        return;
    }
    hijk = sqrt( tmpFloat);
    pijk0 = bijk + uijk * hijk;
    pijk1 = bijk - uijk * hijk;

    bool draw0, draw1;
    draw0 = true;
    draw1 = true;
    
    int stop1 = min( id1, id2);
    int stop2 = max( id1, id2);
    float4 voxel;
    float3 voxelPos;
    for( cutId = 0; cutId < stop1; ++cutId ) {
        voxel = FETCH( atomPos, FETCH( neighbors, params.maxNumNeighbors * atomId + cutId));
        voxelPos = make_float3( voxel.x, voxel.y, voxel.z);
        if( length( pijk0 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw0 = false;
        if( length( pijk1 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw1 = false;
    }
    for( cutId = stop1+1; cutId < stop2; ++cutId ) {
        voxel = FETCH( atomPos, FETCH( neighbors, params.maxNumNeighbors * atomId + cutId));
        voxelPos = make_float3( voxel.x, voxel.y, voxel.z);
        if( length( pijk0 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw0 = false;
        if( length( pijk1 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw1 = false;
    }
    for( cutId = stop2+1; cutId < vicinityCnt; ++cutId ) {
        voxel = FETCH( atomPos, FETCH( neighbors, params.maxNumNeighbors * atomId + cutId));
        voxelPos = make_float3( voxel.x, voxel.y, voxel.z);
        if( length( pijk0 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw0 = false;
        if( length( pijk1 - voxelPos ) < ( params.probeRadius + voxel.w - 0.001f ) )
            draw1 = false;
    }
    
    if( draw0 && draw1 ) {
        point1[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_uint4( aiIdx, ajIdx, visibleAtomIdx, 1);
        probePos[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_float4( pijk0.x, pijk0.y, pijk0.z,-1.0f);
    } else if( draw0 && !draw1 ) {
        point1[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_uint4( aiIdx, ajIdx, visibleAtomIdx, 1);
        probePos[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_float4( pijk0.x, pijk0.y, pijk0.z, 1.0f);
    } else if( !draw0 && draw1 ) {
        point1[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_uint4( aiIdx, ajIdx, visibleAtomIdx, 1);
        probePos[visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX] = make_float4( pijk1.x, pijk1.y, pijk1.z, 1.0f);
    } else {
        return;
    }
    // ---------- ... FRAGMENT SHADER CODE ---------

}

///////////////////////////////////////////////////////////////////////////////
// compute the triangle vbo
///////////////////////////////////////////////////////////////////////////////
__global__
void computeTriangleVBO( float3* vbo,           // output: triangle vertices and colors
                  //float4* point1,               // input: point 1 of the RS face
                  //float4* point2,               // input: point 2 of the RS face
                  //float4* point3 ) {            // input: point 3 of the RS face
                  uint4* point1,                // input: point 1 of the RS face
                  float4* atomPos,              // input: sorted atom positions
                  float4* visibleAtoms,         // input: visible atoms position and radius
                  uint offset ) {

    // get atom index
    uint visibleAtomIdx = offset + blockIdx.y * blockDim.y + threadIdx.y;
    // check bounds of visible atoms
    if( visibleAtomIdx >= rsParams.visibleAtomCount ) {
        return;
    }
    // get combined neighbor index
    uint idxX = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds of neighbor index
    if( idxX >= ( params.maxNumNeighbors * params.maxNumNeighbors) ) {
        return;
    }

    // the color value for visibility checking
    //float3 color = { float( visibleAtomIdx), float( idxX), 0.0f};
    float3 color = { float( idxX), float( visibleAtomIdx), 0.0f};
    float4 pos; 
    // compute the index of the array
    uint pointIdx = visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX;
    // offset: 3 vertices + 3 colors
    //uint vboIdx = pointIdx * 6;
    uint vboIdx = ( ( visibleAtomIdx - offset) * params.maxNumNeighbors * params.maxNumNeighbors + idxX) * 6;
    // write the positions
    //uint4 point = FETCH( point1, pointIdx);
    uint4 point = point1[pointIdx];
    //pos = FETCH( atomPos, point1[pointIdx].x);
    pos = FETCH( atomPos, point.x);
    pos.w = 1.0f;
    vbo[vboIdx+0] = make_float3( pos);

    color.z = 0.0f;
    vbo[vboIdx+1] = color;

    //pos = FETCH( atomPos, point1[pointIdx].y);
    pos = FETCH( atomPos, point.y);
    pos.w = 1.0f;
    vbo[vboIdx+2] = make_float3( pos);

    color.z = 1.0f;
    vbo[vboIdx+3] = color;

    //pos = FETCH( visibleAtoms, point1[pointIdx].z);
    pos = FETCH( visibleAtoms, point.z);
    pos.w = 1.0f;
    vbo[vboIdx+4] = make_float3( pos);

    color.z = 2.0f;
    vbo[vboIdx+5] = color;
}

///////////////////////////////////////////////////////////////////////////////
// compute the triangle vbo
///////////////////////////////////////////////////////////////////////////////
__global__
void computeVisibleTriangleVBO( float3* vbo,           // output: triangle vertices and colors
                                uint4* point1,         // input: point 1 of the RS face
                                float4* atomPos,       // input: sorted atom positions
                                float4* visibleAtoms,  // input: visible atoms position and radius
                                uint offset ) {

    // get atom index
    uint visibleAtomIdx = offset + blockIdx.y * blockDim.y + threadIdx.y;
    // check bounds of visible atoms
    if( visibleAtomIdx >= rsParams.visibleAtomCount ) {
        return;
    }
    // get combined neighbor index
    uint idxX = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds of neighbor index
    if( idxX >= ( params.maxNumNeighbors * params.maxNumNeighbors) ) {
        return;
    }

    // the color value for visibility checking
    //float3 color = { float( visibleAtomIdx), float( idxX), 0.0f};
    float3 color = { float( idxX), float( visibleAtomIdx), 0.0f};
    float4 pos; 
    // compute the index of the array
    uint pointIdx = visibleAtomIdx * params.maxNumNeighbors * params.maxNumNeighbors + idxX;
    // offset: 3 vertices + 3 colors
    //uint vboIdx = pointIdx * 6;
    uint vboIdx = ( ( visibleAtomIdx - offset) * params.maxNumNeighbors * params.maxNumNeighbors + idxX) * 6;
    // get the visiblity
    float visible = tex2D( inVisibilityTex, idxX, visibleAtomIdx).x;
    // write the positions
    //uint4 point = FETCH( point1, pointIdx);
    uint4 point = point1[pointIdx];
    //pos = FETCH( atomPos, point1[pointIdx].x);
    pos = FETCH( atomPos, point.x);
    pos.w = 1.0f;
    vbo[vboIdx+0] = make_float3( pos) * visible;

    color.z = 0.0f;
    vbo[vboIdx+1] = color;

    //pos = FETCH( atomPos, point1[pointIdx].y);
    pos = FETCH( atomPos, point.y);
    pos.w = 1.0f;
    vbo[vboIdx+2] = make_float3( pos) * visible;

    color.z = 1.0f;
    vbo[vboIdx+3] = color;

    //pos = FETCH( visibleAtoms, point1[pointIdx].z);
    pos = FETCH( visibleAtoms, point.z);
    pos.w = 1.0f;
    vbo[vboIdx+4] = make_float3( pos) * visible;

    color.z = 2.0f;
    vbo[vboIdx+5] = color;
}


///////////////////////////////////////////////////////////////////////////////
// compute the torus vbo
///////////////////////////////////////////////////////////////////////////////
__global__
void computeTorusVBO(
        float4* outTorusVBO,    // the output VBO (positions + attributes for torus drawing)
        float4* outSTriaVBO,    // the output VBO (positions + attributes for spherical triangle drawing)
        float4* inVBO,          // the input VBO (indices of visible triangles)
        float4* atomPos,        // the sorted atom positions (for neighboring atoms)
        float4* visibleAtoms,   // the visible atoms' positions
        uint4* point1,          // the atom index array
        float4* probePos ) {    // the probe position array
    // get the index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    // get the coordinates of the atom index array
    float4 indicesFloat = inVBO[idx];
    uint xIdx = uint( floor( indicesFloat.x + 0.5f));
    uint yIdx = uint( floor( indicesFloat.y + 0.5f));
    uint arrayIdx = params.maxNumNeighbors * params.maxNumNeighbors * yIdx + xIdx;
    uint4 pointIdx = point1[arrayIdx];
    // get the points
    float4 aiTmp = FETCH( atomPos, pointIdx.x);
    float3 ai = make_float3( aiTmp.x, aiTmp.y, aiTmp.z);
    float ri = aiTmp.w;
    float4 ajTmp = FETCH( atomPos, pointIdx.y);
    float3 aj = make_float3( ajTmp.x, ajTmp.y, ajTmp.z);
    float rj = ajTmp.w;
    float4 akTmp = FETCH( visibleAtoms, pointIdx.z);
    float3 ak = make_float3( akTmp.x, akTmp.y, akTmp.z);
    float rk = akTmp.w;
    // get the probe position
    float4 probe = probePos[arrayIdx];

    // names of the variables according to: Connolly "Analytical Molecular Surface Calculation", 1983
    float3 tij, tik, tjk, pijk0, pijk1;
    float dij, dik, djk, rij, rik, rjk;

    dij = length( aj - ai);
    dik = length( ak - ai);
    djk = length( ak - aj);
    
    tij = 0.5f*( ai + aj) + 0.5f*( aj - ai) * ( ( ri + params.probeRadius)*( ri + params.probeRadius) - ( rj + params.probeRadius)*( rj + params.probeRadius))/( dij*dij);
    tik = 0.5f*( ai + ak) + 0.5f*( ak - ai) * ( ( ri + params.probeRadius)*( ri + params.probeRadius) - ( rk + params.probeRadius)*( rk + params.probeRadius))/( dik*dik);
    tjk = 0.5f*( aj + ak) + 0.5f*( ak - aj) * ( ( rj + params.probeRadius)*( rj + params.probeRadius) - ( rk + params.probeRadius)*( rk + params.probeRadius))/( djk*djk);
    rij = 0.5f*sqrt( float( (ri + rj + 2.0f*params.probeRadius)*(ri + rj + 2.0f*params.probeRadius) - dij*dij)) * ( sqrt( float( dij*dij - ( ri - rj)*( ri - rj))) / dij);
    rik = 0.5f*sqrt( float( (ri + rk + 2.0f*params.probeRadius)*(ri + rk + 2.0f*params.probeRadius) - dik*dik)) * ( sqrt( float( dik*dik - ( ri - rk)*( ri - rk))) / dik);
    rjk = 0.5f*sqrt( float( (rj + rk + 2.0f*params.probeRadius)*(rj + rk + 2.0f*params.probeRadius) - djk*djk)) * ( sqrt( float( djk*djk - ( rj - rk)*( rj - rk))) / djk);
    pijk0 = make_float3( probe.x, probe.y, probe.z);
    pijk1 = pijk0*probe.w;

    //////////////////////////////////////////////
    // emit varyings and position for torus i-j //
    //////////////////////////////////////////////
    // get the rotation axis of the torus
    float3 torusAxis = normalize( ai - tij);
    // get the axis for rotating the torus rotations axis on the z-axis
    float3 rotAxis = normalize( cross( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    // compute quaternion
    float4 quatC;
    float angle = acos( dot( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    float len = length( rotAxis);
    float halfAngle = 0.5f * angle;
    if( len > 0.0f ) {
        len = sin( halfAngle);
        quatC.x = rotAxis.x * len;
        quatC.y = rotAxis.y * len;
        quatC.z = rotAxis.z * len;
        quatC.w = cos( halfAngle);
    } else {
        quatC = make_float4( 0.0f, 0.0f, 0.0f, 1.0f);
    }
    // compute the tangential point X2 of the spheres
    float3 P = tij + rotAxis * rij;
    float3 X1 = normalize( P - ai) * ri;
    float3 X2 = normalize( P - aj) * rj;
    float3 C = ai - aj;
    C = ( length( P - aj) / ( length( P - ai) + length( P - aj) ) ) * C;
    float distance = length( X2 - C);
    C = ( C + aj) - tij;
    // write the parameters
    outTorusVBO[idx*3*4] = make_float4( tij.x, tij.y, tij.z, 1.0f);
    outTorusVBO[idx*3*4+1] = make_float4( params.probeRadius, rij, 1.0f, 1.0f);
    outTorusVBO[idx*3*4+2] = quatC;
    outTorusVBO[idx*3*4+3] = make_float4( C.x, C.y, C.z, distance);

    //////////////////////////////////////////////
    // emit varyings and position for torus i-k //
    //////////////////////////////////////////////
    // get the rotation axis of the torus
    torusAxis = normalize( ai - tik);
    // get the axis for rotating the torus rotations axis on the z-axis
    rotAxis = normalize( cross( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    // compute quaternion
    angle = acos( dot( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    len = length( rotAxis);
    halfAngle = 0.5f * angle;
    if( len > 0.0f ) {
        len = sin( halfAngle);
        quatC.x = rotAxis.x * len;
        quatC.y = rotAxis.y * len;
        quatC.z = rotAxis.z * len;
        quatC.w = cos( halfAngle);
    } else {
        quatC = make_float4( 0.0f, 0.0f, 0.0f, 1.0f);
    }
    // compute the tangential point X2 of the spheres
    P = tik + rotAxis * rik;
    X1 = normalize( P - ai) * ri;
    X2 = normalize( P - ak) * rk;
    C = ai - ak;
    C = ( length( P - ak) / ( length( P - ai) + length( P - ak) ) ) * C;
    distance = length( X2 - C);
    C = ( C + ak) - tik;
    // write the parameters
    outTorusVBO[idx*3*4+4] = make_float4( tik.x, tik.y, tik.z, 1.0f);
    outTorusVBO[idx*3*4+5] = make_float4( params.probeRadius, rik, 1.0f, 1.0f);
    outTorusVBO[idx*3*4+6] = quatC;
    outTorusVBO[idx*3*4+7] = make_float4( C.x, C.y, C.z, distance);

    //////////////////////////////////////////////
    // emit varyings and position for torus i-k //
    //////////////////////////////////////////////
    // get the rotation axis of the torus
    torusAxis = normalize( aj - tjk);
    // get the axis for rotating the torus rotations axis on the z-axis
    rotAxis = normalize( cross( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    // compute quaternion
    angle = acos( dot( torusAxis, make_float3( 0.0f, 0.0f, 1.0f)));
    len = length( rotAxis);
    halfAngle = 0.5f * angle;
    if( len > 0.0f ) {
        len = sin(halfAngle);
        quatC.x = rotAxis.x * len;
        quatC.y = rotAxis.y * len;
        quatC.z = rotAxis.z * len;
        quatC.w = cos( halfAngle);
    } else {
        quatC = make_float4( 0.0f, 0.0f, 0.0f, 1.0f);
    }
    // compute the tangential point X2 of the spheres
    P = tjk + rotAxis * rjk;
    X1 = normalize( P - aj) * rj;
    X2 = normalize( P - ak) * rk;
    C = aj - ak;
    C = ( length( P - ak) / ( length( P - aj) + length( P - ak) ) ) * C;
    distance = length( X2 - C);
    C = ( C + ak) - tjk;
    // write the parameters
    outTorusVBO[idx*3*4+8] = make_float4( tjk.x, tjk.y, tjk.z, 1.0f);
    outTorusVBO[idx*3*4+9] = make_float4( params.probeRadius, rjk, 1.0f, 1.0f);
    outTorusVBO[idx*3*4+10] = quatC;
    outTorusVBO[idx*3*4+11] = make_float4( C.x, C.y, C.z, distance);

    /////////////////////////////////////////////////////////////
    // emit varyings and position for first spherical triangle //
    /////////////////////////////////////////////////////////////
    outSTriaVBO[idx*2*4] = make_float4( pijk0, params.probeRadius);
    outSTriaVBO[idx*2*4+1] = make_float4( ai - pijk0, 1.0f);
    outSTriaVBO[idx*2*4+2] = make_float4( aj - pijk0, 1.0f);
    outSTriaVBO[idx*2*4+3] = make_float4( ak - pijk0, params.probeRadius*params.probeRadius);
    
    //////////////////////////////////////////////////////////////
    // emit varyings and position for second spherical triangle //
    //////////////////////////////////////////////////////////////
    outSTriaVBO[idx*2*4+4] = make_float4( pijk1, params.probeRadius);
    outSTriaVBO[idx*2*4+5] = make_float4( ai - pijk1, 1.0f);
    outSTriaVBO[idx*2*4+6] = make_float4( aj - pijk1, 1.0f);
    outSTriaVBO[idx*2*4+7] = make_float4( ak - pijk1, params.probeRadius*params.probeRadius);
    
    ///////////////////////////////////////////////////////////////////////
    // ==> The two spherical triangles are potentially the same!
    //     This does not matter, since rendering if fast enough and
    //     singularity handling will test probe distances.
    ///////////////////////////////////////////////////////////////////////
    
}


///////////////////////////////////////////////////////////////////////////////
// write the probe positions to a new array
///////////////////////////////////////////////////////////////////////////////
__global__
void writeProbePositions(
        float4* probePos,   // output (probe positions)
        float4* sTriaVbo,   // input (probe positions)
        uint numProbes ) {  // the number of probes
    // get the index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds
    if( idx >= numProbes ) return;
    // copy data
    probePos[idx] = sTriaVbo[idx*4];
    probePos[idx].w = 1.0f;
}

///////////////////////////////////////////////////////////////////////////////
// write the singularities to the PBO
///////////////////////////////////////////////////////////////////////////////
__global__
void writeSingularities(
        float3* outArray,
        uint*  probeNeighbors,
        float4* probePos ) {
    // get the indices
    uint probeIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // check bounds of visible atoms
    if( probeIdx >= rsParams.probeCount ) {
        return;
    }
    // get combined neighbor index
    uint neighborIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds of neighbor index
    if( neighborIdx >= rsParams.maxNumProbeNeighbors ) {
        return;
    }
    // read the neighbor probe position
    uint nIdx = probeNeighbors[probeIdx*rsParams.maxNumProbeNeighbors+neighborIdx];
    float4 pos = probePos[nIdx];
    // write the neighbor probe position
    outArray[probeIdx*rsParams.maxNumProbeNeighbors+neighborIdx] = make_float3( pos.x, pos.y, pos.z);
}

///////////////////////////////////////////////////////////////////////////////
// find all adjacent, occluded RS-faces
///////////////////////////////////////////////////////////////////////////////
__global__
void findAdjacentTriangles( 
        float* outPbo,
        uint4* point1, 
        float4* probePos,
        uint* neighborCount,
        uint* neighbors,
        float4* atomPos, 
        float4* visibleAtoms, 
        uint* visibleAtomsId ) {
    // get atom index
    uint visibleAtomIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // check bounds of visible atoms
    if( visibleAtomIdx >= rsParams.visibleAtomCount ) {
        return;
    }
    // get neighbor index
    uint nIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds of neighbor index
    if( nIdx >= params.maxNumNeighbors ) {
        return;
    }
    
    uint cnt;
    bool visible = false;
    bool secondVisible = false;
    float4 visibleTria[3];
    float4 visibleTriaNormal;
    float4 invisibleTriaNormal[64];
    uint invisibleTriaId[64];
    float4 tmp;
    float3 tmpDir, tmpDirTS;
    int smallest = -1;

    uint arrayIdx;
    uint4 pointIdx;

    uint counter = 0;
    
    int xcoord = int( nIdx)*params.maxNumNeighbors;

    float visibility;

    // check number of visible triangles for this edge
    for( cnt = 0; cnt < params.maxNumNeighbors; ++cnt ) {
        // compute the array index
        arrayIdx = params.maxNumNeighbors * params.maxNumNeighbors * visibleAtomIdx + xcoord + cnt;
        // get the indices of the atoms
        pointIdx = point1[arrayIdx];
        // get visibility information
        visibility = tex2D( inVisibilityTex, xcoord+cnt, visibleAtomIdx).x;
        // copy visibility information to PBO
        outPbo[arrayIdx] = visibility;
        // check visibility
        if( tex2D( inVisibilityTex, xcoord+cnt, visibleAtomIdx).x > 0.5f ) {
            // if a second visible triangle was found: do nothing!
            if( visible ) secondVisible = true;
            // get the points
            visibleTria[0] = FETCH( atomPos, pointIdx.x);
            visibleTria[1] = FETCH( atomPos, pointIdx.y);
            visibleTria[2] = FETCH( visibleAtoms, pointIdx.z);
            // get the probe position
            visibleTriaNormal = probePos[arrayIdx];
            visible = true;
        } else {
            tmp = FETCH( atomPos, pointIdx.y);
            if( tmp.w > 0.5f ) {
                invisibleTriaNormal[counter] = probePos[arrayIdx];
                invisibleTriaId[counter] = cnt;
                counter++;
            }
        }
    }

    // if no or two visible triangles were found: do nothing!
    if( !visible || secondVisible ) return;
    
    float angle, tmpAngle;
    // The transformation matrix is:
    //     (Tx Ty Tz 0)
    // M = (Bx By Bz 0)
    //     (Nx Ny Nz 0)
    //     ( 0  0  0 1)
    // where T is the tangent, B is binormal and N is the normal (all in object space).
    // T = shared edge ai-ak; B = pijk - tik; N = T x B
    float3 ai = make_float3( visibleTria[0].x, visibleTria[0].y, visibleTria[0].z);
    float ri = visibleTria[0].w;
    float3 aj = make_float3( visibleTria[1].x, visibleTria[1].y, visibleTria[1].z);
    float3 ak = make_float3( visibleTria[2].x, visibleTria[2].y, visibleTria[2].z);
    float rk = visibleTria[2].w;

    float3 T = normalize( ai - ak);

    float dik = length( ak - ai);
    float3 tik = 0.5f*( ai + ak) + 0.5f*( ak - ai) * ( ( ri + params.probeRadius)*( ri + params.probeRadius) - ( rk + params.probeRadius)*( rk + params.probeRadius))/( dik*dik);
    float3 B = normalize( make_float3( visibleTriaNormal.x, visibleTriaNormal.y, visibleTriaNormal.z) - tik);
    
    float3 N = normalize( cross( T, B));
    
    // set angle to more than 2*PI
    angle = 7.0f;
    for( cnt = 0; cnt < counter; ++cnt ) {
        // get direction to pijk'
        tmpDir = normalize( make_float3( invisibleTriaNormal[cnt].x, invisibleTriaNormal[cnt].y, invisibleTriaNormal[cnt].z) - tik);
        // project direction to tangent space
        tmpDirTS = make_float3( dot(T, tmpDir), dot(B, tmpDir), dot(N, tmpDir));
        tmpAngle = atan2f( tmpDirTS.z, tmpDirTS.y) + 3.14159265f;
        if( tmpAngle < angle ) {
            angle = tmpAngle;
            smallest = invisibleTriaId[cnt];
        }
        // get direction to pijk'2
        tmpDir = normalize( make_float3( invisibleTriaNormal[cnt].x, invisibleTriaNormal[cnt].y, invisibleTriaNormal[cnt].z)*invisibleTriaNormal[cnt].w - tik);
        // project direction to tangent space
        tmpDirTS = make_float3( dot(T, tmpDir), dot(B, tmpDir), dot(N, tmpDir));
        tmpAngle = atan2f( tmpDirTS.z, tmpDirTS.y) + 3.14159265f;
        if( tmpAngle < angle ) {
            angle = tmpAngle;
            smallest = invisibleTriaId[cnt];
        }
    }
    
    if( smallest >= 0 )
        outPbo[params.maxNumNeighbors * params.maxNumNeighbors * visibleAtomIdx + xcoord + uint(smallest)] = 1.0f;
}

///////////////////////////////////////////////////////////////////////////////
//
// CONTOUR BUILDUP KERNELS
//
///////////////////////////////////////////////////////////////////////////////

// find all neighbor atoms in a given cell
__device__ uint findNeighborsInCellCBCuda(
        uint*   neighbors,      // out: neighbor indices
        float4* smallCircles,   // out: small circles
        uint    neighborIndex,  // in: first index for writing in neighbor list
        int3    gridPos,        // in: the current grid cell
        uint    index,          // in: the index of the atom
        float4  pos,            // in: the position of the atom
        float4* atomPos,        // in: the (sorted) atom position array
        uint*   cellStart,
        uint*   cellEnd) {
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH( cellStart, gridHash);

    uint count = 0;
    float4 pos2;
    float3 relPos;
    float dist;
    float neighborDist;
    float r;
    float3 vec;
    float4 smallCircle;
    if( startIndex != 0xffffffff ) {	// cell is not empty
        // iterate over atoms in this cell
        uint endIndex = FETCH( cellEnd, gridHash);
        for( uint j = startIndex; j < endIndex; j++) {
            // do not count self
            if( j != index) {
                // get position of potential neighbor
                pos2 = FETCH( atomPos, j);
                // check distance
                relPos = make_float3( pos2.x, pos2.y, pos2.z) - make_float3( pos.x, pos.y, pos.z);
                dist = length( relPos);
                neighborDist = pos.w + pos2.w + 2.0f * params.probeRadius;
                if( dist < neighborDist ) {
                    // check number of neighbors
                    if( ( neighborIndex + count) >= params.maxNumNeighbors ) return count;
                    // write the (sorted) neighbor index
                    neighbors[index*params.maxNumNeighbors+neighborIndex+count] = j;
                    // compute small circle / intersection plane
                    r = ( (pos.w + params.probeRadius)*(pos.w + params.probeRadius))
                        + ( dist * dist)
                        - ( (pos2.w + params.probeRadius)*(pos2.w + params.probeRadius));
                    r = r / (2.0f * dist * dist);
                    vec = relPos * r;
                    // set small circle
                    smallCircle.x = vec.x;
                    smallCircle.y = vec.y;
                    smallCircle.z = vec.z;
                    //smallCircle.w = 1.0f;
                    smallCircle.w = sqrt(((pos.w + params.probeRadius) * (pos.w + params.probeRadius)) - dot( vec, vec));
                    smallCircles[index*params.maxNumNeighbors+neighborIndex+count] = smallCircle;
                    // increment the neighbor counter
                    count++;
                }
            }
        }
    }
    return count;
}

// Find and count neighbors and write them to the neighbor array.
// Also computes small circles for each neighbor and stores them, too.
__global__ void findNeighborsCBCuda(
        uint*   neighborCount,     // out: number of neighbors
        uint*   neighbors,         // out: neighbor indices
        float4* smallCircles,      // out: small circles
        float4* atomPos,           // in: sorted atom positions
        uint*   cellStart,
        uint*   cellEnd,
        uint    numAtoms) {
    uint index = __mul24( blockIdx.x, blockDim.x) + threadIdx.x;
    if( index >= numAtoms ) return;

    // read atom data from sorted arrays
    float4 pos = FETCH( atomPos, index);

    // get address in grid
    int3 gridPos = calcGridPos( make_float3( pos));

    int3 gridSize;
    gridSize.x = int( params.gridSize.x);
    gridSize.y = int( params.gridSize.y);
    gridSize.z = int( params.gridSize.z);
    // search range for neighbor atoms: max atom diameter + probe diameter
    float range = ( pos.w + 3.0f + 2.0f * params.probeRadius);
    // compute number of grid cells
    int3 cellsInRange;
    cellsInRange.x = ceil( range / params.cellSize.x);
    cellsInRange.y = ceil( range / params.cellSize.y);
    cellsInRange.z = ceil( range / params.cellSize.z);
    int3 start = gridPos - cellsInRange;
    int3 end = gridPos + cellsInRange;

    // examine neighbouring cells
    uint count = 0;
    int3 neighborPos;
    for( int z = ( start.z > 0 ? start.z : 0); z < ( end.z > gridSize.z ? gridSize.z : end.z) ; z++ ) {
        for( int y = ( start.y > 0 ? start.y : 0); y < ( end.y > gridSize.y ? gridSize.y : end.y) ; y++ ) {
            for( int x = ( start.x > 0 ? start.x : 0); x < ( end.x > gridSize.x ? gridSize.x : end.x) ; x++ ) {
                neighborPos = make_int3( x, y, z);
                count += findNeighborsInCellCBCuda( neighbors, smallCircles, count, neighborPos, index, pos, atomPos, cellStart, cellEnd);
            }
        }
    }

    // write new neighbor atom count back to (sorted) index location
    neighborCount[index] = count;
}

// find and remove unnecessary small circles
__global__ void removeCoveredSmallCirclesCBCuda(
        float4* smallCircles,         // in/out: small circles
        uint*   smallCircleVisible,   // out: small circle visibility
        uint*   neighborCount,        // in/out: number of neighbors
        uint*   neighbors,            // input: neighbor indices
        float4* atomPos,              // input: sorted atom positions
        uint    numAtoms) {
    // get atom index
    uint atomIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // get neighbor atom index
    uint jIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // check, if atom index is within bounds
    if( atomIdx >= numAtoms ) return;
    // check, if neighbor index is within bounds
    if( jIdx >= params.maxNumNeighbors ) return;
    // set small circle visibility to false
    smallCircleVisible[atomIdx * params.maxNumNeighbors + jIdx] = 0;
    // check, if neighbor index is within bounds
    uint numNeighbors = neighborCount[atomIdx];
    if( jIdx >= numNeighbors ) return;

    // read position and radius of atom i from sorted array
    float4 atomi = FETCH( atomPos, atomIdx);
    float3 pi = make_float3( atomi.x, atomi.y, atomi.z);
    float R = atomi.w + params.probeRadius;

    // flag wether j sould be added (true) is cut off (false)
    bool addJ = true;

    // the atom index of j
    uint j = FETCH( neighbors, atomIdx * params.maxNumNeighbors + jIdx);
    // get small circle j
    float4 scj = smallCircles[atomIdx * params.maxNumNeighbors + jIdx];
    // vj = the small circle center
    float3 vj = make_float3( scj.x, scj.y, scj.z);
    // pj = center of atom j
    float4 aj = FETCH( atomPos, j);
    float3 pj = make_float3( aj.x, aj.y, aj.z);
    // variables for k
    uint k;
    float4 sck;
    float3 vk;
    float4 ak;
    float3 pk;
    float vjvk;
    float denom;
    float3 h;
    float3 nj;
    float3 nk;
    float3 q;
    float3 mj;
    float3 mk;

    // check j with all other neighbors k
    for( uint kCnt = 0; kCnt < numNeighbors; kCnt++ ) {
        // don't compare the circle with itself
        if( jIdx != kCnt ) {
            // the atom index of k
            k = FETCH( neighbors, atomIdx * params.maxNumNeighbors + kCnt);
            // pk = center of atom k
            ak = FETCH( atomPos, k);
            pk = make_float3( ak.x, ak.y, ak.z);
            // get small circle k
            sck = smallCircles[atomIdx * params.maxNumNeighbors + kCnt];
            // vk = the small circle center
            vk = make_float3( sck.x, sck.y, sck.z);
            // vj * vk
            vjvk = dot( vj, vk);
            // denominator
            denom = dot( vj, vj) * dot( vk, vk) - vjvk * vjvk;
            // point on straight line (intersection of small circle planes)
            h = vj * ( dot( vj, vj - vk) * dot( vk, vk) ) / denom + vk * ( dot( vk - vj, vk) * dot( vj, vj) ) / denom;
            // compute cases
            nj = normalize( pi - pj);
            nk = normalize( pi - pk);
            q = vk - vj;
            // if normals are the same (unrealistic, yet theoretically possible)
            if( dot( nj, nk) == 1.0f ) {
                if( dot( nj, nk) > 0.0f ) {
                    if( dot( nj, q) > 0.0f ) {
                        // k cuts off j --> remove j
                        addJ = false;
                    }
                }
            } else if( length( h) > R ) {
                mj = ( vj - h);
                mk = ( vk - h);
                if( dot( nj, nk) > 0.0f ) {
                    if( dot( mj, mk) > 0.0f && dot( nj, q) > 0.0f ) {
                        // k cuts off j --> remove j
                        addJ = false;
                    }
                } else {
                    if( dot( mj, mk) > 0.0f && dot( nj, q) < 0.0f ) {
                        // atom i has no contour
                        neighborCount[atomIdx] = 0;
                    }
                }
            }
        }
    }
    // all k were tested, see if j is cut off
    if( !addJ ) {
        smallCircles[atomIdx * params.maxNumNeighbors + jIdx].w = -1.0f;
    }
}

// compute all arcs of atom j on the surface of atom i
__global__ void computeArcsCBCuda(
        uint*   smallCircleVisible,		// out: small circle visibility
        float4* arcs,					// out: the arcs
        uint*   arcCount,				// out: the number of arcs
        uint    numAtoms) {
    // get atom index
    uint atomIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // get neighbor atom index
    uint jIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // check, if atom index is within bounds
    if( atomIdx >= numAtoms ) return;
    // check, if neighbor index is within bounds
    if( jIdx >= params.maxNumNeighbors ) return;
    // check, if neighbor index is within bounds
    uint numNeighbors = FETCH( neighborCount, atomIdx);
    if( jIdx >= numNeighbors ) return;

    // read position and radius of atom i from sorted array
    float4 atomi = FETCH( atomPos, atomIdx);
    float3 pi = make_float3( atomi.x, atomi.y, atomi.z);
    float R = atomi.w + params.probeRadius;

    // the atom index of j
    //uint j = neighbors[atomIdx * params.maxNumNeighbors + jIdx];
    uint j = FETCH( neighbors, atomIdx * params.maxNumNeighbors + jIdx);
    // get small circle j
    float4 scj = FETCH( smallCircles, atomIdx * params.maxNumNeighbors + jIdx);
    // do nothing if small circle j has radius -1 (removed)
    if( scj.w < 0.0f )
        return;
    // vj = the small circle center
    float3 vj = make_float3( scj.x, scj.y, scj.z);
    // pj = center of atom j
    float4 aj = FETCH( atomPos, j);
    float3 pj = make_float3( aj.x, aj.y, aj.z);
    // store all arcs
    float start[64];
    float end[64];
    uint startkIndex[64];
    uint endkIndex[64];
    bool arcValid[64];
    start[0] = 0.0f;
    end[0] = 6.28318530718f;
    startkIndex[0] = 0;
    endkIndex[0] = 0;
    arcValid[0] = true;
    uint arcCnt = 1;
    // temporary arc arrays for new arcs
    float tmpStart[16];
    float tmpEnd[16];
    uint tmpStartkIndex[16];
    uint tmpEndkIndex[16];
    uint tmpArcCnt = 0;
    // compute axes of local coordinate system
    float3 ex = make_float3( 1.0f, 0.0f, 0.0f);
    float3 ey = make_float3( 0.0f, 1.0f, 0.0f);
    float3 xAxis = cross( vj, ey);
    if( dot( xAxis, xAxis) == 0.0f ) {
        xAxis = cross( vj, ex);
    }
    xAxis = normalize( xAxis);
    float3 yAxis = cross( xAxis, vj);
    yAxis = normalize( yAxis);

    // variables for k
    uint k;
    float4 sck;
    float3 vk;
    float4 ak;
    float3 pk;
    float vjvk;
    float denom;
    float3 h;
    float root;
    float3 x1;
    float3 x2;
    float3 tmpVec;
    float xX1;
    float yX1;
    float xX2;
    float yX2;
    float angleX1;
    float angleX2;
    uint aCnt;
    float s;
    float e;
    float skIndex;
    float ekIndex;

    // check j with all other neighbors k
    for( uint kCnt = 0; kCnt < numNeighbors; kCnt++ ) {
        // don't compare the circle with itself
        if( jIdx == kCnt ) 
            continue;
        // the atom index of k
        //k = neighbors[atomIdx * params.maxNumNeighbors + kCnt];
        k = FETCH( neighbors, atomIdx * params.maxNumNeighbors + kCnt);
        // get small circle k
        sck = FETCH( smallCircles, atomIdx * params.maxNumNeighbors + kCnt);
        // do nothing if small circle k has radius -1 (removed)
        if( sck.w < 0.0f )
            continue;
        // vk = the small circle center
        vk = make_float3( sck.x, sck.y, sck.z);
        // pk = center of atom k
        ak = FETCH( atomPos, k);
        pk = make_float3( ak.x, ak.y, ak.z);
        // vj * vk
        vjvk = dot( vj, vk);
        // denominator
        denom = dot( vj, vj) * dot( vk, vk) - vjvk * vjvk;
        // point on straight line (intersection of small circle planes)
        h = vj * ( dot( vj, vj - vk) * dot( vk, vk) ) / denom + vk * ( dot( vk - vj, vk) * dot( vj, vj) ) / denom;

        // do nothing if h is outside of the extended sphere of atom i
        if( length( h) > R ) 
            continue;
        // compute the root
        root = sqrt( ( R*R - dot( h, h)) / dot( cross( vk, vj), cross( vk, vj)));
        // compute the two intersection points
        x1 = h + cross( vk, vj) * root;
        x2 = h - cross( vk, vj) * root;
        // swap x1 & x2 if vj points in the opposit direction of pj-pi
        if( dot( vk, pk - pi) < 0.0f ) {
            tmpVec = x1;
            x1 = x2;
            x2 = tmpVec;
        }
        // transform x1 and x2 to small circle coordinate system
        xX1 = dot( x1 - vj, xAxis);
        yX1 = dot( x1 - vj, yAxis);
        xX2 = dot( x2 - vj, xAxis);
        yX2 = dot( x2 - vj, yAxis);
        angleX1 = atan2( yX1, xX1);
        angleX2 = atan2( yX2, xX2);
        // limit angles to 0..2*PI
        if( angleX1 > 6.28318530718f ) {
            angleX1 = fmod( angleX1, 6.28318530718f);
            angleX2 = fmod( angleX2, 6.28318530718f);
        }
        // angle of x2 has to be larger than angle of x1 (add 2 PI)
        if( angleX2 < angleX1 ) {
            angleX2 += 6.28318530718f;
        }
        // make all angles positive (add 2 PI)
        if( angleX1 < 0.0f ) {
            angleX1 += 6.28318530718f;
            angleX2 += 6.28318530718f;
        }

        // check all existing arcs with new arc k
        for( aCnt = 0; aCnt < arcCnt; aCnt++ ) {
            s = start[aCnt];
            e = end[aCnt];
            skIndex = startkIndex[aCnt];
            ekIndex = endkIndex[aCnt];
            if( arcValid[aCnt] ) {
                if( angleX1 < s ) {
                    // case (1) & (10)
                    if( ( s - angleX1) > ( angleX2 - angleX1)) {
                        if( ( ( s - angleX1) + ( e - s)) > 6.28318530718f ) {
                            if( ( ( s - angleX1) + ( e - s)) < ( 6.28318530718f + angleX2 - angleX1) ) {
                                // case (10)
                                start[aCnt] = angleX1;
                                startkIndex[aCnt] = k;
                                end[aCnt] = fmod( e, 6.28318530718f);
                                // second angle check
                                if( end[aCnt] < start[aCnt] )
                                    end[aCnt] += 6.28318530718f;
                            } else {
                                start[aCnt] = angleX1;
                                startkIndex[aCnt] = k;
                                end[aCnt] = fmod( angleX2, 6.28318530718f);
                                endkIndex[aCnt] = k;
                                // second angle check
                                if( end[aCnt] < start[aCnt] )
                                    end[aCnt] += 6.28318530718f;
                            }
                        } else {
                            // case (1)
                            //arcAngles.RemoveAt( aCnt);
                            //aCnt--;
                            arcValid[aCnt] = false;
                        }
                    } else {
                        if( ( ( s - angleX1) + ( e - s)) > ( angleX2 - angleX1) ) {
                            // case (5)
                            end[aCnt] = fmod( angleX2, 6.28318530718f);
                            endkIndex[aCnt] = k;
                            // second angle check
                            if( end[aCnt] < start[aCnt] )
                                end[aCnt] += 6.28318530718f;
                            if( ( ( s - angleX1) + ( e - s)) > 6.28318530718f ) {
                                // case (6)
                                tmpStart[tmpArcCnt] = angleX1;
                                tmpStartkIndex[tmpArcCnt] = k;
                                tmpEnd[tmpArcCnt] = fmod( e, 6.28318530718f);
                                tmpEndkIndex[tmpArcCnt] = ekIndex;
                                // second angle check
                                if( tmpEnd[tmpArcCnt] < tmpStart[tmpArcCnt] )
                                    tmpEnd[tmpArcCnt] += 6.28318530718f;
                                tmpArcCnt++;
                            }
                        }
                    } // case (4): Do nothing!
                } else { // angleX1 > s
                    // case (2) & (9)
                    if( ( angleX1 - s) > ( e - s)) {
                        if( ( ( angleX1 - s) + ( angleX2 - angleX1)) > 6.28318530718f ) {
                            if( ( ( angleX1 - s) + ( angleX2 - angleX1)) < ( 6.28318530718f + e - s)) {
                                // case (9)
                                end[aCnt] = fmod( angleX2, 6.28318530718f);
                                endkIndex[aCnt] = k;
                                // second angle check
                                if( end[aCnt] < start[aCnt] )
                                    end[aCnt] += 6.28318530718f;
                            }
                        } else {
                            // case (2)
                            //arcAngles.RemoveAt( aCnt);
                            //aCnt--;
                            arcValid[aCnt] = false;
                        }
                    } else {
                        if( ( ( angleX1 - s) + ( angleX2 - angleX1)) > ( e - s) ) {
                            // case (7)
                            start[aCnt] = angleX1;
                            startkIndex[aCnt] = k;
                            // second angle check
                            end[aCnt] = fmod( end[aCnt], 6.28318530718f);
                            if( end[aCnt] < start[aCnt] )
                                end[aCnt] += 6.28318530718f;
                            if( ( ( angleX1 - s) + ( angleX2 - angleX1)) > 6.28318530718f ) {
                                // case (8)
                                tmpStart[tmpArcCnt] = s;
                                tmpStartkIndex[tmpArcCnt] = skIndex;
                                tmpEnd[tmpArcCnt] = fmod( angleX2, 6.28318530718f);
                                tmpEndkIndex[tmpArcCnt] = k;
                                // second angle check
                                if( tmpEnd[tmpArcCnt] < tmpStart[tmpArcCnt] )
                                    tmpEnd[tmpArcCnt] += 6.28318530718f;
                                tmpArcCnt++;
                            }
                        } else {
                            // case (3)
                            start[aCnt] = angleX1;
                            startkIndex[aCnt] = k;
                            end[aCnt] = fmod( angleX2, 6.28318530718f);
                            endkIndex[aCnt] = k;
                            // second angle check
                            if( end[aCnt] < start[aCnt] )
                                end[aCnt] += 6.28318530718f;
                        }
                    }
                }
            } // if( arcValid[aCnt] )
        } // for( uint aCnt = 0; aCnt < arcCnt; aCnt++ )

        // copy new arcs to arc array
        for( aCnt = 0; aCnt < tmpArcCnt; aCnt++ ) {
            start[aCnt + arcCnt] = tmpStart[aCnt];
            end[aCnt + arcCnt] = tmpEnd[aCnt];
            startkIndex[aCnt + arcCnt] = tmpStartkIndex[aCnt];
            endkIndex[aCnt + arcCnt] = tmpEndkIndex[aCnt];
            arcValid[aCnt + arcCnt] = true;
        }
        // add new arcs to arc count
        arcCnt += tmpArcCnt;
        // "reset" temporary arc array
        tmpArcCnt = 0;

        // fill gaps (overwrite invalid arcs)
        uint counter = 0;
        for( aCnt = 0; aCnt < arcCnt; aCnt++ ) {
            if( arcValid[aCnt] ) {
                start[aCnt - counter] = start[aCnt];
                end[aCnt - counter] = end[aCnt];
                startkIndex[aCnt - counter] = startkIndex[aCnt];
                endkIndex[aCnt - counter] = endkIndex[aCnt];
                arcValid[aCnt - counter] = arcValid[aCnt];
            } else {
                counter++;
            }
        }
        // subtract number of invalid arcs from total number of arcs
        arcCnt -= counter;
    } // for( uint kCnt = 0; kCnt < numNeighbors; kCnt++ )

    // TODO: remove/merge split arcs ( x..2*PI / 0..y --> x..y+2*PI )

    // merge arcs if arc with angle 0 and arc with angle 2*PI exist
    int idx0 = -1;
    int idx2pi = -1;
    for( aCnt = 0; aCnt < arcCnt; aCnt++ ) {
        if( start[aCnt] < 0.00001f ) {
            idx0 = int( aCnt);
        } else if( abs( end[aCnt] - 6.28318530718f) < 0.0001f ) {
            idx2pi = int( aCnt);
        }
    }
    if( idx0 >= 0 && idx2pi >= 0 ) {
        start[uint(idx0)] = start[uint(idx2pi)];
        startkIndex[uint(idx0)] = startkIndex[uint(idx2pi)];
        // second angle check
        end[uint(idx0)] = fmod( end[uint(idx0)], 6.28318530718f);
        if( end[uint(idx0)] < start[uint(idx0)] )
            end[uint(idx0)] += 6.28318530718f;
        // fill gaps (overwrite removed arc idx2pi)
        for( aCnt = uint(idx2pi); aCnt < arcCnt - 1; aCnt++ ) {
            start[aCnt] = start[aCnt + 1];
            end[aCnt] = end[aCnt + 1];
            startkIndex[aCnt] = startkIndex[aCnt + 1];
            endkIndex[aCnt] = endkIndex[aCnt + 1];
            arcValid[aCnt] = true;
        }
        // subtract the removed arc from total number of arcs
        arcCnt--;
    }

    uint arcWritten = 0;
    // copy arcs to global arc array
    for( aCnt = 0; aCnt < arcCnt; aCnt++ ) {
        if( atomIdx < j ) {
            if( j < startkIndex[aCnt] ) {
                arcs[atomIdx * params.maxNumNeighbors * params.maxNumNeighbors + jIdx * params.maxNumNeighbors + arcWritten] = 
                    make_float4( pi + vj + ( cos( start[aCnt]) * xAxis + sin( start[aCnt]) * yAxis) * scj.w, float( startkIndex[aCnt]) + 0.2f); //start[aCnt]);
                arcWritten++;
            }
            if( j < endkIndex[aCnt] ) {
                arcs[atomIdx * params.maxNumNeighbors * params.maxNumNeighbors + jIdx * params.maxNumNeighbors + arcWritten] = 
                    make_float4( pi + vj + ( cos( end[aCnt]) * xAxis + sin( end[aCnt]) * yAxis) * scj.w, float( endkIndex[aCnt]) + 0.2f); //end[aCnt]);
                arcWritten++;
            }
        }
    }

    // write number of arcs
    arcCount[atomIdx * params.maxNumNeighbors + jIdx] = arcWritten;

    // set small circle j visible if at least one arc was created and i < j
    if( atomIdx < j && arcCnt > 0 ) {
    //if( arcWritten > 0 ) {
        smallCircleVisible[atomIdx * params.maxNumNeighbors + jIdx] = 1;
    }
    // DO NOT USE THIS!! It will create false, internal arcs!
    //if( arcCnt == 0 ) {
    //	smallCircles[atomIdx * params.maxNumNeighbors + jIdx].w = -1.0f;
    //}

}

// Write all arc start and end points to a vertex array.
__global__ void writeProbePositionsCBCuda(
        float4*	probePos,		// out: the probe positions
        float4*	sphereTriaVec1,	// out: the spherical triangle vector 1
        float4*	sphereTriaVec2,	// out: the spherical triangle vector 2
        float4*	sphereTriaVec3,	// out: the spherical triangle vector 3
        float4*	torusPos,		// out: torus center
        float4*	torusVS,		// out: torus visibility sphere
        float4*	torusAxis,		// out: torus axis
        float4* arcs,			// in: the arc start and end points
        uint*	arcCount,		// in: the number of probes
        uint*	arcCountScan,	// in: the prefix sum of "arcCount"
        uint*	scCount,		// in: the small circle count
        uint*	scCountScan,	// in: the prefix sum of "scCount"
        uint    numAtoms) {
    // get atom index
    uint atomIdx = blockIdx.y * blockDim.y + threadIdx.y;
    // get neighbor atom index
    uint jIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // check, if atom index is within bounds
    if( atomIdx >= numAtoms ) return;
    // check, if neighbor index is within bounds
    uint numNeighbors = FETCH( neighborCount, atomIdx);
    if( jIdx >= numNeighbors ) return;

    float4 ai = FETCH( atomPos, atomIdx);
    float4 aj = FETCH( atomPos, FETCH( neighbors, atomIdx * params.maxNumNeighbors + jIdx));
    float4 ak;

    // ---------- write spherical triangle ----------

    // get number of probes and the sum of previous probes for this neighbor
    uint numProbes = arcCount[atomIdx * params.maxNumNeighbors + jIdx];
    uint numPreviousProbes = arcCountScan[atomIdx * params.maxNumNeighbors + jIdx];

    float4 tmpProbePos;
    uint kIdx;

    for( uint cnt = 0; cnt < numProbes; cnt++ ) {
        tmpProbePos = arcs[atomIdx * params.maxNumNeighbors * params.maxNumNeighbors + jIdx * params.maxNumNeighbors + cnt];
        probePos[numPreviousProbes + cnt] = make_float4( tmpProbePos.x, tmpProbePos.y, tmpProbePos.z, params.probeRadius);
        kIdx = uint( floor( tmpProbePos.w));
        //ak = FETCH( atomPos, FETCH( neighbors, kIdx));
        ak = FETCH( atomPos, kIdx);
        sphereTriaVec1[numPreviousProbes + cnt] = make_float4( make_float3( ai) - make_float3( tmpProbePos), 1.0f);
        sphereTriaVec2[numPreviousProbes + cnt] = make_float4( make_float3( aj) - make_float3( tmpProbePos), 1.0f);
        sphereTriaVec3[numPreviousProbes + cnt] = make_float4( make_float3( ak) - make_float3( tmpProbePos), params.probeRadius * params.probeRadius);
    }

    // ---------- write torus ----------

    if( scCount[atomIdx * params.maxNumNeighbors + jIdx] > 0 ) {
        float4 sc;
        uint torusIdx = scCountScan[atomIdx * params.maxNumNeighbors + jIdx];
        sc = FETCH( smallCircles, atomIdx * params.maxNumNeighbors + jIdx);
        // torus axis
        float3 ta = normalize( make_float3( sc));
        // torus center
        float3 tc = make_float3( sc) + make_float3( ai);
        float3 ortho = normalize( cross( ta, make_float3( 0.0f, 0.0f, 1.0f)));

        // compute the tangential point X2 of the spheres
        float3 P = tc + ( ortho * sc.w);
        float3 X = normalize( P - make_float3( ai)) * ai.w;
        float3 C = ( length( P - make_float3( ai)) /
            ( length( P - make_float3( aj)) + length( P - make_float3( ai)))) * 
            ( make_float3( aj) - make_float3( ai));
        float distance = length( X - C);
        C = ( C + make_float3( ai)) - tc;

        // write torus center & torus radius R
        torusPos[torusIdx] = make_float4( tc, sc.w);
        // write torus axis & probe radius (= torus radius r)
        torusAxis[torusIdx] = make_float4( ta, params.probeRadius);
        // write visibility sphere
        torusVS[torusIdx] = make_float4( C, distance);
    }

}

// find all probe neighbors in a given cell
__device__ uint findProbeNeighborsInCellCBCuda(
        float3* singTex,        // out: singularity texture
        uint    neighborIndex,  // in: first index for writing in neighbor list
        int3    gridPos,        // in: the current grid cell
        uint    index,          // in: the index of the probe
        float4  pos,            // in: the position of the probe
        float4* probePos,       // in: the (sorted) probe position array
        uint*   gridProbeIndex, // input: sorted probe indices
        uint*   cellStart,
        uint*   cellEnd,
        uint    numNeighbors ) {
    uint gridHash = calcGridHash(gridPos);

    // get start of bucket for this cell
    uint startIndex = FETCH( cellStart, gridHash);

    uint count = 0;
    float4 pos2;
    float3 relPos;
    float dist;
    float neighborDist;
    if( startIndex != 0xffffffff ) {	// cell is not empty
        // iterate over atoms in this cell
        uint endIndex = FETCH( cellEnd, gridHash);
        for( uint j = startIndex; j < endIndex; j++) {
            // do not count self
            if( j != index) {
                // get position of potential neighbor
                pos2 = probePos[j];
                // check distance
                relPos = make_float3( pos2.x, pos2.y, pos2.z) - make_float3( pos.x, pos.y, pos.z);
                dist = length( relPos);
                neighborDist = params.probeRadius + params.probeRadius;
                if( dist < neighborDist ) {
                    // check number of neighbors
                    if( ( neighborIndex + count) >= numNeighbors ) return count;
                    // write singTex
                    singTex[index * numNeighbors + neighborIndex + count] = make_float3( pos2);
                    // increment the neighbor counter
                    count++;
                }
            }
        }
    }
    return count;
}

// Find and count probe neighbors and write them to the singularity texture
__global__ void findProbeNeighborsCBCuda(
        float3* texCoord,       // out: #neighbors + texture coords
        float3* singTex,        // out: singularity texture
        float4* probePos,       // in: probe positions
        uint*   gridProbeIndex, // input: sorted probe indices
        uint*   cellStart,
        uint*   cellEnd,
        uint    numProbes,
        uint    numNeighbors ) {
    uint index = __mul24( blockIdx.x, blockDim.x) + threadIdx.x;
    if( index >= numProbes ) return;

    // read atom data from sorted arrays
    float4 pos = probePos[index];

    // get address in grid
    int3 gridPos = calcGridPos( make_float3( pos));

    int3 gridSize;
    gridSize.x = int( params.gridSize.x);
    gridSize.y = int( params.gridSize.y);
    gridSize.z = int( params.gridSize.z);
    // search range for neighbor atoms: max atom diameter + probe diameter
    float range = params.probeRadius + params.probeRadius;
    // compute number of grid cells
    int3 cellsInRange;
    cellsInRange.x = ceil( range / params.cellSize.x);
    cellsInRange.y = ceil( range / params.cellSize.y);
    cellsInRange.z = ceil( range / params.cellSize.z);
    int3 start = gridPos - cellsInRange;
    int3 end = gridPos + cellsInRange;

    // examine neighbouring cells
    uint count = 0;
    int3 neighborPos;
    for( int z = ( start.z > 0 ? start.z : 0); z < ( end.z > gridSize.z ? gridSize.z : end.z) ; z++ ) {
        for( int y = ( start.y > 0 ? start.y : 0); y < ( end.y > gridSize.y ? gridSize.y : end.y) ; y++ ) {
            for( int x = ( start.x > 0 ? start.x : 0); x < ( end.x > gridSize.x ? gridSize.x : end.x) ; x++ ) {
                neighborPos = make_int3( x, y, z);
                count += findProbeNeighborsInCellCBCuda( singTex, count, neighborPos, index, pos, probePos, gridProbeIndex, cellStart, cellEnd, numNeighbors);
            }
        }
    }

    uint xCoord = index / params.texSize;
    // write probe neighbor count and texture coordinates
    texCoord[gridProbeIndex[index]] = make_float3( float( count), float( xCoord), float( index % params.texSize));
}

#endif
