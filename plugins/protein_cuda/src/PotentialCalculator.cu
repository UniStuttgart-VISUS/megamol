#include "hip/hip_runtime.h"
//
// PotentialCalculator.cu
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on: Apr 23, 2013
//     Author: scharnkn
//

#include "gpu_poisson_solver.cu" // Note: by Georg Rempfer (georg@icp.uni-stuttgart.de)
#include "cuenergy.cu"
#include "cuda_error_check.h"
#ifdef _WIN64
#include "hipfft/hipfft.h"

#define CUERR { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return err; }}

typedef unsigned int uint;


extern "C"
hipError_t SolvePoissonEq(float gridSpacing, uint3 gridSize, float *charges,
        float *potential_D, float *potential) {

    unsigned int Nx = gridSize.x;
    unsigned int Ny = gridSize.y;
    unsigned int Nz = gridSize.z;
    float h = gridSpacing; // Gridspacing

    printf("Calculating electrostatic potential on a %d*%d*%d grid with spacing %f\n", Nx, Ny, Nz, h);

    /* timing */
    float time = 0.0, time_tmp;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipfftHandle plan_fft;
    hipfftHandle plan_ifft;
    hipfftComplex* data_dev;
    hipfftComplex* data_host;
    hipfftReal* data_real_host;
    hipfftReal* greensfunc_dev;
    hipfftReal* greensfunc_host;

    hipMalloc((void**) &data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1));

    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return hipGetLastError();
    }

    hipMalloc((void**) &greensfunc_dev, sizeof(hipfftReal)*Nz*Ny*(Nx/2+1));

    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return hipGetLastError();
    }

    hipHostMalloc((void**) &data_host, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1));

    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return hipGetLastError();
    }

    data_real_host = (hipfftReal*) data_host;

    hipHostMalloc((void**) &greensfunc_host, sizeof(hipfftReal)*Nz*Ny*(Nx/2+1));

    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to allocate\n");
        return hipGetLastError();
    }

    /* greens function */
    printf("Creating greens function in device memory\n");

    createGreensFunc <<< 14, 32*32 >>> (greensfunc_dev, Nx, Ny, Nz, h);

    /* charge density */
    printf("Writing charge density in host memory\n");

	for (int z = 0; z < (int)Nz; ++z) {
		for (int y = 0; y < (int)Ny; ++y) {
			for (int x = 0; x < (int)Nx; ++x) {
//                if((x-Nx/2)*(x-Nx/2) + (y-Ny/2)*(y-Ny/2) + (z-Nz/2)*(z-Nz/2) <= 5*5/(h*h)) //homogeneously chargeed sphere of radius 5
//                    data_real_host[Ny*Nx*z+Nx*y+x] = h*h*h;
//                else
//                    data_real_host[Ny*Nx*z+Nx*y+x] = 0.0;
                data_real_host[Ny*Nx*z+Nx*y+x] = charges[Ny*Nx*z+Nx*y+x];
            }
        }
    }

    printf("Copying charge density to device\n");

    hipMemcpy(data_dev, data_host, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyHostToDevice);

    if(hipGetLastError() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to copy\n");
        return hipGetLastError();
    }

    /* create 3D FFT plans */
    printf("Setting up FFT and iFFT plans\n");

    /* Notice how the directions x and z are exchanged. This is because for R2C
       transforms, cuda only stores half the results in the 3rd direction. At
       the same time cuda expects the fastest running index to be the one with
       only half the values stored, which effectively forces one to make the 3rd
       index (usually z) the fastest running one. I find this rather uncommon
       and want x to be the festest running index and z the slowest running, so
       I chose to exchange the two in the fourier transforms. */
    if(hipfftPlan3d(&plan_fft, Nz, Ny, Nx, HIPFFT_R2C) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to create fft plan\n");
        return hipGetLastError();
    }

    /*if(cufftSetCompatibilityMode(plan_fft, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to set fft compatibility mode to native\n");
        return hipGetLastError();
    }*/

    if(hipfftPlan3d(&plan_ifft, Nz, Ny, Nx, HIPFFT_C2R) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to create ifft plan\n");
        return hipGetLastError();
    }

    /*if(cufftSetCompatibilityMode(plan_ifft, CUFFT_COMPATIBILITY_NATIVE) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to set ifft compatibility mode to native\n");
        return hipGetLastError();
    }*/

    /* FFT in place */
    printf("Executing FFT in place\n");

    hipEventRecord(start, 0);

    if(hipfftExecR2C(plan_fft, (hipfftReal*) data_dev, data_dev) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to execute FFT plan\n");
        return hipGetLastError();
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;

    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return hipGetLastError();
    }

    /* multiplying with greens function */
    printf("Executing multiplication with greens function in place\n");

    hipEventRecord(start, 0);

    //18-fold occupation seems to be optimal for the GT520 and 32-fold for the C2050
    multiplyGreensFunc <<<14,32*32>>> (data_dev,
            greensfunc_dev, Nz*Ny*(Nx/2+1));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;

    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return hipGetLastError();
    }


    /* inverse FFT in place */
    printf("Executing iFFT in place\n");

    hipEventRecord(start, 0);

    if(hipfftExecC2R(plan_ifft, data_dev, (hipfftReal*) data_dev) != HIPFFT_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to execute iFFT plan\n");
        return hipGetLastError();
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tmp, start, stop);
    printf("Execution time: %f ms\n", time_tmp);
    time += time_tmp;

    if(hipDeviceSynchronize() != hipSuccess) {
        fprintf(stderr, "ERROR: Failed to synchronize\n");
        return hipGetLastError();
    }

    /* retrieving result from device */
    hipMemcpy(data_host, data_dev, sizeof(hipfftComplex)*Nz*Ny*(Nx/2+1), hipMemcpyDeviceToHost);

    /* Output result to the according host array */

	for (int z = 0; z < (int)Nz; z++) {
		for (int y = 0; y < (int)Ny; y++)
			for (int x = 0; x < (int)Nx; x++)
               potential[Ny*Nx*z+Nx*y+x]  = data_real_host[Ny*Nx*z+Nx*y+x]/(Nx*Ny*Nz);
    }

    /* cleanup */
    printf("Cleanup\n");

    hipfftDestroy(plan_fft);
    hipfftDestroy(plan_ifft);

    hipFree(data_dev);
    hipFree(greensfunc_dev);
    hipFree(data_real_host); // TODO This causes invalif device pointer error, why?
    hipFree(greensfunc_host);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Net device execution time: %f ms\n", time);

    return hipGetLastError();
}

extern "C"
hipError_t DirectCoulombSummation(float *atomData, uint atomCount,
        float *potential, uint3 gridSize, float gridspacing) {

    // TODO Why is this so incredibely slow??

    float *doutput = NULL;
    dim3 volsize, Gsz, Bsz;
    //float copytotal, runtotal, mastertotal, hostcopytotal;
    const char *statestr = "|/-\\.";
    int state=0;

    printf("CUDA accelerated coulombic potential microbenchmark V4.0\n");
    printf("John E. Stone <johns@ks.uiuc.edu>\n");
    printf("and Chris Rodrigues\n");
    printf("http://www.ks.uiuc.edu/Research/gpu/\n");
    printf("--------------------------------------------------------\n");
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Detected %d CUDA accelerators:\n", deviceCount);
    int dev;
    for (dev=0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        printf("  [%d]: '%s'  Clock: %.1f GHz  Mem: %dMB  Rev: %d.%d\n",
                dev, deviceProp.name,
                deviceProp.clockRate / 1000000.0f, deviceProp.totalGlobalMem / (1024*1024),
                deviceProp.major, deviceProp.minor);
    }

    int cudadev = 0; // Use first cuda device
    //      if (argc == 2) {
    //        sscanf(argv[1], "%d", &cudadev);
    //        if (cudadev < 0 || cudadev >= deviceCount) {
    //          cudadev = 0;
    //        }
    //      }
    printf("  Single-threaded single-GPU test run.\n");
    printf("  Opening CUDA device %d...\n", cudadev);
    hipSetDevice(cudadev);
    CUERR // check and clear any existing errors TODO

    // number of atoms to simulate
    int atomcount = 100000;

    // setup energy grid size
    // XXX this is a large test case to clearly illustrate that even while
    //     the CUDA kernel is running entirely on the GPU, the CUDA runtime
    //     library is soaking up the entire host CPU for some reason.
    volsize = gridSize;

    // setup CUDA grid and block sizes
    // XXX we have to make a trade-off between the number of threads per
    //     block and the resulting padding size we'll end up with since
    //     each thread will do several consecutive grid cells in this version,
    //     we're using up some of our available parallelism to reduce overhead.
    Bsz.x = BLOCKSIZEX;
    Bsz.y = BLOCKSIZEY;
    Bsz.z = 1;
    Gsz.x = max(1, volsize.x / (Bsz.x * UNROLLX));
    Gsz.y = max(1, volsize.y / (Bsz.y * UNROLLY));
    Gsz.z = volsize.z;

//    printf("Run the kernel, gridSize %u %u %u, blockSize %u %u %u\n", (Bsz.x * UNROLLX),
//            Gsz.y, Gsz.z, Bsz.x, Bsz.y, Bsz.z);

    // allocate and initialize the GPU output array
    int volmemsz = sizeof(float) * volsize.x * volsize.y * volsize.z;
    printf("Allocating %.2fMB of memory for output buffer...\n", volmemsz / (1024.0 * 1024.0));

    hipMalloc((void**)&doutput, volmemsz);
    CUERR // check and clear any existing errors TODO
    hipMemset(doutput, 0, volmemsz);
    CUERR // check and clear any existing errors TODO

    for (uint z = 0; z < volsize.z; ++z) {
        printf("starting run for slab %u...\n", z);

        int iterations=0;
        int atomstart;

        for (atomstart = 0; atomstart < atomcount; atomstart += MAXATOMS) {

            iterations++;
            int runatoms;
            int atomsremaining = atomcount - atomstart;
            if (atomsremaining > MAXATOMS)
                runatoms = MAXATOMS;
            else
                runatoms = atomsremaining;

            printf("%c\r", statestr[state]);
            fflush(stdout);
            state = (state+1) & 3;

            // copy the atoms to the GPU

            if (copyatomstoconstbuf(atomData + 4*atomstart, runatoms, z*gridspacing))
                return hipGetLastError();

            CUERR // check and clear any existing errors

            // RUN the kernel...
            cenergy <<< Gsz, Bsz >>> (runatoms, gridspacing, doutput);

            // TODO
            CUERR // check and clear any existing errors
        }
        hipDeviceSynchronize();
    }

    // Copy the GPU output data back to the host and use/store it..
    hipMemcpy(potential, doutput, volmemsz,  hipMemcpyDeviceToHost);



    //TODO
    CUERR // check and clear any existing errors

#if 1
    int x, y;
    for (y=0; y<16; y++) {
        for (x=0; x<16; x++) {
            int addr = y * volsize.x + x;
            printf("out[%d]: %f\n", addr, potential[addr]);
        }
    }
#endif

//    printf("Final calculation required %d iterations of %d atoms\n", iterations, MAXATOMS);
//    printf("Copy time: %f seconds, %f per iteration\n", copytotal, copytotal / (float) iterations);
//    printf("Kernel time: %f seconds, %f per iteration\n", runtotal, runtotal / (float) iterations);
//    printf("Total time: %f seconds\n", mastertotal);
//    printf("Kernel invocation rate: %f iterations per second\n", iterations / mastertotal);
//    printf("GPU to host copy bandwidth: %gMB/sec, %f seconds total\n",
//            (volmemsz / (1024.0 * 1024.0)) / hostcopytotal, hostcopytotal);

//    double atomevalssec = ((double) volsize.x * volsize.y * volsize.z * atomcount) / (mastertotal * 1000000000.0);
//    printf("Efficiency metric, %g billion atom evals per second\n", atomevalssec);
//
//    /* 59/8 FLOPS per atom eval */
//    printf("FP performance: %g GFLOPS\n", atomevalssec * (59.0/8.0));

    hipFree(doutput);

    return hipGetLastError();
}



#endif // _WIN64
