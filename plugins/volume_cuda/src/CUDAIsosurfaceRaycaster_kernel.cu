#include "hip/hip_runtime.h"
#ifndef VOLUME_CUDA_CUDAISOSURFACERAYCASTER_KERNEL_H_INCLUDED
#define VOLUME_CUDA_CUDAISOSURFACERAYCASTER_KERNEL_H_INCLUDED

#include "CUDAIsosurfaceRaycaster_kernel.cuh"
#include "CUDAGenericFunctions.cuh"

using namespace megamol;
using namespace megamol::volume_cuda;

#include <thrust/extrema.h>
#include <thrust/device_ptr.h>

/**
 * CUDAIsosurfaceRaycaster_kernel::d_render
 */
__global__ void d_renderIso(uint * d_output, float * d_depth, hipTextureObject_t tex, hipTextureObject_t transferTex,
    float * d_isovalues, int numisovalues,
    uint imageW, uint imageH, float fovx, float fovy,
    float3 camPos, float3 camDir, float3 camUp, float3 camRight, float zNear, float zFar,
    float density, float brightness, float transferOffset, float transferScale, float minVal, float maxVal,
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f), const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f),
    hipExtent volSize = make_hipExtent(1, 1, 1)) {

	int maxSteps = 450;

	float tstep = (boxMax.x - boxMin.x) / static_cast<float>(maxSteps);
	if ((boxMax.y - boxMin.y) / static_cast<float>(maxSteps) > tstep) {
		tstep = (boxMax.y - boxMin.y) / static_cast<float>(maxSteps);
	}
	if ((boxMax.z - boxMin.z) / static_cast<float>(maxSteps) > tstep) {
		tstep = (boxMax.z - boxMin.z) / static_cast<float>(maxSteps);
	}
	maxSteps *= 2; // security factor, could be sqrt(2)

	const float opacityThreshold = 0.95f;

	// pixel coordinates
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	if ((x >= imageW) || (y >= imageH)) return;

	// read the depth value and transform it to world coordinates
	float dv = 1.0f;
	if (d_depth != NULL) {
		dv = d_depth[y * imageW + x];
	}

	// TODO correct depth value
	//float depthVal = (2.0f * zNear) / (zFar + zNear - dv * (zFar - zNear));
	//depthVal = zNear + depthVal * (zFar - zNear);

	//printf("%f %f\n", dv, depthVal);

	// texture coordinates
	float u = (x / static_cast<float>(imageW)) * 2.0f - 1.0f;
	float v = (y / static_cast<float>(imageH)) * 2.0f - 1.0f;

	// calculate intersection with near plane in world space
	float3 oL = (tan(fovx * 0.5f) * zNear) * (-camRight) + (tan(fovy * 0.5) * zNear) * camUp + camDir * zNear + camPos;
	float3 uL = (tan(fovx * 0.5f) * zNear) * (-camRight) + (tan(fovy * 0.5) * zNear) * (-camUp) + camDir * zNear + camPos;
	float3 oR = (tan(fovx * 0.5f) * zNear) * camRight + (tan(fovy * 0.5) * zNear) * camUp + camDir * zNear + camPos;
	float3 uR = (tan(fovx * 0.5f) * zNear) * camRight + (tan(fovy * 0.5) * zNear) * (-camUp) + camDir * zNear + camPos;

	float3 targetL = lerp(uL, oL, (v + 1.0f) * 0.5f);
	float3 targetR = lerp(uR, oR, (v + 1.0f) * 0.5f);

	float3 target = lerp(targetL, targetR, (u + 1.0f) * 0.5f);

	// calculate eye ray in world space
	Ray eyeRay;
	eyeRay.o = camPos;
	eyeRay.d = normalize(target - camPos);

	// find intersection with box
	float tnear, tfar;
	int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
	if (!hit) {
		d_output[y*imageW + x] = rgbaFloatToInt(make_float4(0.0f));
		return;
	}

	if (tnear < 0.0f) tnear = 0.0f; // clamp to near plane

	// march along ray from front to back, accumulating colour
	float4 sum = make_float4(0.0f);
	float t = tnear;
	float3 pos = eyeRay.o + eyeRay.d * tnear;
	float3 step = eyeRay.d * tstep;
	float3 diff = boxMax - boxMin;

    if (numisovalues < 1) {
        d_output[y*imageW + x] = rgbaFloatToInt(make_float4(0.0f));
        return;
    }

    float3 sP;
    sP.x = (pos.x - boxMin.x) / diff.x;
    sP.y = (pos.y - boxMin.y) / diff.y;
    sP.z = (pos.z - boxMin.z) / diff.z;
    float val = (tex3D<float>(tex, sP.x, sP.y, sP.z) - minVal / (maxVal - minVal));

    //float isoDiff = 0;
    //float isoDiffOld = val - d_isovalues[0];

    float * isoDiffs = new float[numisovalues];
    float * isoDiffsOld = new float[numisovalues];

    for (int i = 0; i < numisovalues; i++) {
        isoDiffs[i] = 0.0f;
        isoDiffsOld[i] = val - d_isovalues[i];
    }

    float3 voxelSize = make_float3(1.0f / (float)volSize.width, 1.0f / (float)volSize.height, 1.0f / (float)volSize.depth);

    //float alpha = 1.0f / (float)numisovalues;

    float4 colors = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
    // TODO make colors changeable

    //bool firstHit = true;
    //float3 firstHitPos;

	float projA = -(zFar + zNear) / (zFar - zNear);
	float projB = -2.0f * zNear * zFar / (zFar - zNear);

	for (int i = 0; i < maxSteps; i++) {
		// remap position to [0, 1] coordinates
		float3 samplePos;
		samplePos.x = (pos.x - boxMin.x) / diff.x;
		samplePos.y = (pos.y - boxMin.y) / diff.y;
		samplePos.z = (pos.z - boxMin.z) / diff.z;

		// read from 3D texture
		float sample = tex3D<float>(tex, samplePos.x, samplePos.y, samplePos.z);
		
		// normalize the sample
		sample = (sample - minVal) / (maxVal - minVal);

        for (int isoIndex = 0; isoIndex < numisovalues; isoIndex++) {
            isoDiffs[isoIndex] = sample - d_isovalues[isoIndex];

            if ((isoDiffs[isoIndex] * isoDiffsOld[isoIndex] <= 0.0f)) {
                // interpolated exact position of the isosurface
                float3 isoPos = lerp(pos - step, pos, isoDiffsOld[isoIndex] / (isoDiffsOld[isoIndex] - isoDiffs[isoIndex]));
                
                //if (firstHit) {
                //    firstHitPos = isoPos;
                //    firstHit = false;
                //}

                float3 isoSamplePos;
                isoSamplePos.x = (isoPos.x - boxMin.x) / diff.x;
                isoSamplePos.y = (isoPos.y - boxMin.y) / diff.y;
                isoSamplePos.z = (isoPos.z - boxMin.z) / diff.z;

                float3 gradient = make_float3(1, 0, 0);
                gradient.x = ((tex3D<float>(tex, isoSamplePos.x + voxelSize.x, isoSamplePos.y, isoSamplePos.z) - minVal) / (maxVal - minVal))
                    - ((tex3D<float>(tex, isoSamplePos.x - voxelSize.x, isoSamplePos.y, isoSamplePos.z) - minVal) / (maxVal - minVal));
                gradient.y = ((tex3D<float>(tex, isoSamplePos.x, isoSamplePos.y + voxelSize.y, isoSamplePos.z) - minVal) / (maxVal - minVal))
                    - ((tex3D<float>(tex, isoSamplePos.x, isoSamplePos.y - voxelSize.y, isoSamplePos.z) - minVal) / (maxVal - minVal));
                gradient.z = ((tex3D<float>(tex, isoSamplePos.x, isoSamplePos.y, isoSamplePos.z + voxelSize.z) - minVal) / (maxVal - minVal))
                    - ((tex3D<float>(tex, isoSamplePos.x, isoSamplePos.y, isoSamplePos.z - voxelSize.z) - minVal) / (maxVal - minVal));
                gradient = normalize(gradient);

                float4 col = make_float4(0.0);

                col = colors;

                // TODO make this adjustable
                float3 lightDir = make_float3(0.0f, 0.0f, 1.0f);
                float4 lightParams = make_float4(0.1f, 0.3f, 0.2f, 10.0f);
                float4 mycol = CUDAIsosurfaceRaycaster_kernel::performLighting(gradient, -eyeRay.d, lightDir, col, lightParams);

                mycol *= mycol.w;
                mycol.w = col.w;

                sum = sum + (mycol * (1.0f - sum.w));

                // exit early if opaque
                if (sum.w > opacityThreshold) {
                    break;
                }
            }
            isoDiffsOld[isoIndex] = isoDiffs[isoIndex];
        }

		float sampleCamDist = length(eyeRay.o - pos);

		// depth correction if another image is already present
		float localdepth = 0.5f * (-projA * sampleCamDist + projB) / sampleCamDist + 0.5f;
		if (localdepth >= dv) {
			break;
		}

		// lookup in transfer function texture
		float4 col = tex1D<float4>(transferTex, (sample - transferOffset) * transferScale);

		// pre-multiply alpha
		col.x *= col.w;
		col.y *= col.w;
		col.z *= col.w;

		// "over" operator for front-to-back blending
		sum = sum + col * (1.0f - sum.w);

		t += tstep;
		if (t > tfar) break;
		pos += step;
	}
    
	sum *= brightness;

	// write output color
	d_output[y * imageW + x] = rgbaFloatToInt(sum);

    free(isoDiffs);
    free(isoDiffsOld);
}

/**
 * CUDAIsosurfaceRaycaster_kernel::performLighting
 */
__device__ float4 CUDAIsosurfaceRaycaster_kernel::performLighting(float3 normal, float3 camDirection, float3 lightDirection, float4 surfaceColor, float4 lightParams) {
    return make_float4(1.0f, 0.0f, 0.0f, 1.0f);
}
 
/**
 * CUDAIsosurfaceRaycaster_kernel::freeCudaBuffers
 */
void CUDAIsosurfaceRaycaster_kernel::freeCudaBuffers(void) {

    if (d_volumeArray) {
        checkCudaErrors(hipDestroyTextureObject(this->texObj));
        checkCudaErrors(hipFreeArray(d_volumeArray));
    }

    if (d_customTransferFuncArray) {
        checkCudaErrors(hipDestroyTextureObject(this->customTransferTexObj));
        checkCudaErrors(hipFreeArray(d_customTransferFuncArray));
    }
}

/**
 * CUDAIsosurfaceRaycaster_kernel::render_kernel
 */
void CUDAIsosurfaceRaycaster_kernel::render_kernel(dim3 gridSize, dim3 blockSize, uint * d_output, float * d_depth, uint imageW, uint imageH, float fovx, float fovy, float3 camPos, float3 camDir,
	float3 camUp, float3 camRight, float zNear, float zFar, float density, float brightness, float transferOffset, float transferScale,
	const float3 boxMin, const float3 boxMax, hipExtent volSize) {

    int numIsovalues = static_cast<int>(this->isovalues.size());
    float * isovalptr = thrust::raw_pointer_cast(this->isovalues.data());

	d_renderIso <<<gridSize, blockSize >>>(d_output, d_depth, this->texObj, this->customTransferTexObj, isovalptr, numIsovalues, imageW, imageH, fovx, fovy, camPos, camDir, camUp, camRight, zNear, zFar, density, brightness,
		transferOffset, transferScale, minVal, maxVal, boxMin, boxMax, volSize);
}


/**
 * CUDAIsosurfaceRaycaster_kernel::copyColorValues
 */
void CUDAIsosurfaceRaycaster_kernel::copyColorValues(float4 * cvals, int functionSize) {
    if (this->d_customTransferFuncArray) {
        checkCudaErrors(hipDestroyTextureObject(this->customTransferTexObj));
        checkCudaErrors(hipFreeArray(this->d_customTransferFuncArray));
        this->d_customTransferFuncArray = 0;
    }

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
	checkCudaErrors(hipMallocArray(&this->d_customTransferFuncArray, &channelDesc, functionSize, 1));
	checkCudaErrors(hipMemcpyToArray(this->d_customTransferFuncArray, 0, 0, cvals, sizeof(float4)*functionSize, hipMemcpyHostToDevice));

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = this->d_customTransferFuncArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&this->customTransferTexObj, &texRes, &texDescr, NULL));
}

/**
 * CUDAIsosurfaceRaycaster_kernel::transferNewVolume
 */
void CUDAIsosurfaceRaycaster_kernel::transferNewVolume(void * h_volume, hipExtent volumeSize) {
	if (d_volumeArray) {
        checkCudaErrors(hipDestroyTextureObject(this->texObj));
		checkCudaErrors(hipFreeArray(this->d_volumeArray));
		d_volumeArray = 0;
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	checkCudaErrors(hipMalloc3DArray(&this->d_volumeArray, &channelDesc, volumeSize));

	// compute min and max values of the volume
	float * volptr = static_cast<float*>(h_volume);
	thrust::pair<float*, float*> res = thrust::minmax_element(volptr, volptr + (volumeSize.width * volumeSize.depth * volumeSize.height));
	this->minVal = (float)*res.first;
	this->maxVal = (float)*res.second;

	// copy the data
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	copyParams.dstArray = this->d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&copyParams));

    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = this->d_volumeArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;
    texDescr.readMode = hipReadModeElementType;

    checkCudaErrors(hipCreateTextureObject(&this->texObj, &texRes, &texDescr, NULL));
}


/**
 * CUDAIsosurfaceRaycaster_kernel::initCudaDevice
 */
void CUDAIsosurfaceRaycaster_kernel::initCudaDevice(void * h_volume, hipExtent volumeSize, float4 * transferFunction, int functionSize) {
	transferNewVolume(h_volume, volumeSize);
	//copyTransferFunction(transferFunction, functionSize);
}

/**
 * CUDAIsosurfaceRaycaster_kernel::initCudaDevice
 */
void CUDAIsosurfaceRaycaster_kernel::setIsoValues(const std::vector<float>& h_isovalues, int h_numisovalues) {
    this->isovalues = h_isovalues;
}

/**
 * CUDAIsosurfaceRaycaster_kernel::CUDAIsosurfaceRaycaster_kernel
 */
CUDAIsosurfaceRaycaster_kernel::CUDAIsosurfaceRaycaster_kernel(void) {
    this->d_volumeArray = 0;
    this->d_customTransferFuncArray = 0;
    this->texObj = 0;
    this->customTransferTexObj = 0;
}

/**
 * CUDAIsosurfaceRaycaster_kernel::~CUDAIsosurfaceRaycaster_kernel
 */
CUDAIsosurfaceRaycaster_kernel::~CUDAIsosurfaceRaycaster_kernel(void) {
}

#endif /* defined VOLUME_CUDA_CUDAISOSURFACERAYCASTER_KERNEL_H_INCLUDED */